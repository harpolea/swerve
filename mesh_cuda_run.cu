#include "hip/hip_runtime.h"
/**
File containing cuda_run.
**/

void cuda_run(float * beta,
         float ** Us_h, float * rho, float * Q,
         int * nxs, int * nys, int * nzs, int nlevels, char * models,
         int * vec_dims, int ng,
         int nt, float alpha0, float R, float gamma, float E_He, float Cv,
         float zmin,
         float dx, float dy, float dz, float dt, bool burning,
         bool periodic, int dprint, char * filename, char * param_filename,
         MPI_Comm comm, MPI_Status status, int rank, int n_processes,
         int * matching_indices, int r, int n_print_levels,
         int * print_levels, int tstart, float * p_const) {
    /**
    Evolve system through nt timesteps, saving data to filename every dprint timesteps.

    Parameters
    ----------
    beta : float *
        shift vector at each grid point
    gamma_up : float *
        gamma matrix at each grid point
    Us_h : float **
        state vector at each grid point in each layer at current timestep on host in grids
    rho : float *
        densities in each layer
    Q : float *
        heating rate at each point and in each layer
    nxs, nys, nzs : int *
        dimensions of grids
    ng : int
        number of ghost cells
    nt : int
        total number of timesteps
    alpha0 : float
        lapse function at sea floor
    R : float
        radius of star
    gamma : float
        adiabatic index
    E_He : float
        energy release per unit mass of helium
    Cv : float
        specific heat in constant volume
    zmin : float
        height of sea floor
    dx, dy, dz, dt : float
        gridpoint spacing and timestep spacing
    periodic : bool
        do we use periodic or outflow boundary conditions?
    burning : bool
        is burning included in this system?
    dprint : int
        number of timesteps between each printout
    filename : char *
        name of file to which output is printed
    comm : MPI_Comm
        MPI communicator
    status: MPI_Status
        status of MPI processes
    rank, n_processes : int
        rank of current MPI process and total number of MPI processes
    matching_indices : int *
        position of fine grid wrt coarse grid
    r : int
        ratio of resolutions
    print_level : int
        number of the level to be output to file
    */

    // set up GPU stuff
    int count;
    hipGetDeviceCount(&count);

    if (rank == 0) {
        hipError_t err = hipGetLastError();
        // check that we actually have some GPUS
        if (err != hipSuccess) {
            printf("Error: %s\n", hipGetErrorString(err));
            printf("Aborting program.\n");
            return;
        }
        printf("Found %i CUDA devices\n", count);
    }

    // if rank > number of GPUs, exit now
    if (rank >= count) return;

    // redefine - we only want to run on as many cores as we have GPUs
    if (n_processes > count) n_processes = count;

    if (rank == 0) {
        printf("Running on %i processor(s)\n", n_processes);
    }

    int maxThreads = 256;
    int maxBlocks = 160;

    dim3 *kernels = new dim3[n_processes];
    int *cumulative_kernels = new int[n_processes];

    getNumKernels(array_max(nxs, nlevels), array_max(nys, nlevels),
                  array_max(nzs, nlevels), ng, n_processes,
                  &maxBlocks, &maxThreads, kernels, cumulative_kernels);

    int total_kernels = cumulative_kernels[n_processes-1];

    dim3 *blocks = new dim3[total_kernels];
    dim3 *threads = new dim3[total_kernels];

    getNumBlocksAndThreads(array_max(nxs, nlevels), array_max(nys, nlevels),
                           array_max(nzs, nlevels), ng, maxBlocks, maxThreads,
                           n_processes, kernels, blocks, threads);

    printf("rank: %i\n", rank);
    printf("kernels: (%i, %i)\n", kernels[rank].x, kernels[rank].y);
    printf("cumulative kernels: %i\n", cumulative_kernels[rank]);

    int k_offset = 0;
    if (rank > 0) {
      k_offset = cumulative_kernels[rank-1];
    }

    for (int i = k_offset; i < cumulative_kernels[rank]; i++) {
        printf("blocks: (%i, %i, %i) , threads: (%i, %i, %i)\n",
               blocks[i].x, blocks[i].y, blocks[i].z,
               threads[i].x, threads[i].y, threads[i].z);
    }

    // gpu variables
    float * rho_d, * Q_d;

    // set device
    hipSetDevice(rank);

    // index of first multilayer SWE grid level
    int m_in = 0;
    while (models[m_in] != 'M') m_in += 1;
    // index of first compressible grid level
    int c_in = nlevels;
    if (models[nlevels-1] == 'C') {
        while(models[c_in-1] == 'C') c_in -= 1;
    }
    // allocate memory on device
    hipMalloc((void**)&rho_d, nzs[m_in]*sizeof(float));
    hipMalloc((void**)&Q_d, nzs[m_in]*sizeof(float));

    // copy stuff to GPU
    hipMemcpyToSymbol(HIP_SYMBOL(beta_d), beta, 3*sizeof(float));
    hipMemcpy(rho_d, rho, nzs[m_in]*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Q_d, Q, nzs[m_in]*sizeof(float), hipMemcpyHostToDevice);

    int grid_size = nxs[0]*nys[0]*nzs[0]*vec_dims[0];
    for (int i = 1; i < nlevels; i++) {
        grid_size = max(nxs[i]*nys[i]*nzs[i]*vec_dims[i], grid_size);
    }

    int * nxs_d, * nys_d, * nzs_d;
    hipMalloc((void**)&nxs_d, nlevels*sizeof(int));
    hipMalloc((void**)&nys_d, nlevels*sizeof(int));
    hipMalloc((void**)&nzs_d, nlevels*sizeof(int));
    hipMemcpy(nxs_d, nxs, nlevels*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(nys_d, nys, nlevels*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(nzs_d, nzs, nlevels*sizeof(int), hipMemcpyHostToDevice);

    float * U_h = new float[grid_size];
    float * Up_h = new float[grid_size];
    float * F_h = new float[grid_size];

    // initialise
    for (int i = 0; i < grid_size; i++) {
        U_h[i] = 0.0;
        Up_h[i] = 0.0;
        F_h[i] = 0.0;
    }

    float * U_d, * U_half_d, * Up_d, * F_d;

    hipMalloc((void**)&U_d, grid_size*sizeof(float));
    hipMalloc((void**)&U_half_d, grid_size*sizeof(float));
    hipMalloc((void**)&Up_d, grid_size*sizeof(float));
    hipMalloc((void**)&F_d, grid_size*sizeof(float));

    // initialise with coarsest grid
    for (int i = 0; i < nxs[0]*nys[0]*nzs[0]*vec_dims[0]; i++) {
        U_h[i] = Us_h[0][i];
    }
    hipMemcpy(U_d, U_h, nxs[0]*nys[0]*nzs[0]*vec_dims[0]*sizeof(float),
               hipMemcpyHostToDevice);

    hipMemcpy(Up_d, Up_h, grid_size*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(F_d, F_h, grid_size*sizeof(float), hipMemcpyHostToDevice);

    float *qx_p_d, *qx_m_d, *qy_p_d, *qy_m_d, *qz_p_d, *qz_m_d, *fx_p_d,
          *fx_m_d, *fy_p_d, *fy_m_d, *fz_p_d, *fz_m_d;

    hipMalloc((void**)&qx_p_d, grid_size*sizeof(float));
    hipMalloc((void**)&qx_m_d, grid_size*sizeof(float));
    hipMalloc((void**)&qy_p_d, grid_size*sizeof(float));
    hipMalloc((void**)&qy_m_d, grid_size*sizeof(float));
    hipMalloc((void**)&qz_p_d, grid_size*sizeof(float));
    hipMalloc((void**)&qz_m_d, grid_size*sizeof(float));
    hipMalloc((void**)&fx_p_d, grid_size*sizeof(float));
    hipMalloc((void**)&fx_m_d, grid_size*sizeof(float));
    hipMalloc((void**)&fy_p_d, grid_size*sizeof(float));
    hipMalloc((void**)&fy_m_d, grid_size*sizeof(float));
    hipMalloc((void**)&fz_p_d, grid_size*sizeof(float));
    hipMalloc((void**)&fz_m_d, grid_size*sizeof(float));

    // find size of largest compressible and SWE grids
    int largest_comp_grid = 0;
    int largest_swe_grid = 0;
    for (int i = 0; i < nlevels; i++) {
        if (models[i] == 'M' || models[i] == 'S') {
            largest_swe_grid = max(largest_swe_grid, nxs[i]*nys[i]*nzs[i]);
        } else if (models[i] == 'C') {
            largest_comp_grid = max(largest_comp_grid, nxs[i]*nys[i]*nzs[i]);
        }
    }

    float * q_comp_d;
    hipMalloc((void**)&q_comp_d, largest_swe_grid*6*sizeof(float));
    float * qf_swe;
    hipMalloc((void**)&qf_swe, largest_comp_grid*4*sizeof(float));
    float *old_phi_d, *sum_phs_d;
    hipMalloc((void**)&old_phi_d, largest_swe_grid*sizeof(float));
    hipMalloc((void**)&sum_phs_d, largest_swe_grid*sizeof(float));
    float * p_const_d;
    hipMalloc((void**)&p_const_d, nzs[m_in]*sizeof(float));
    hipMemcpy(p_const_d, p_const,
               nzs[m_in]*sizeof(float), hipMemcpyHostToDevice);

    // initialise old_phi with phi on coarsest multilayer SWE grid
    float *pphi = new float[largest_swe_grid];
    for (int j = 0; j < nxs[m_in]*nys[m_in]*nzs[m_in]; j++) {
        pphi[j] = Us_h[m_in][j*vec_dims[m_in]];
    }
    hipMemcpy(old_phi_d, pphi, nxs[m_in]*nys[m_in]*nzs[m_in]*sizeof(float),
               hipMemcpyHostToDevice);

    float * sum_phs_h = new float[largest_swe_grid];

    int * matching_indices_d;
    hipMalloc((void**)&matching_indices_d, (nlevels-1)*4*sizeof(int));
    hipMemcpy(matching_indices_d, matching_indices,
               (nlevels-1)*4*sizeof(int), hipMemcpyHostToDevice);

    // make host-side function pointers to __device__ functions
    flux_func_ptr h_compressible_fluxes;
    flux_func_ptr h_shallow_water_fluxes;

    // copy function pointers to host equivalent
    hipMemcpyFromSymbol(&h_compressible_fluxes, HIP_SYMBOL(d_compressible_fluxes),
                         sizeof(flux_func_ptr));
    hipMemcpyFromSymbol(&h_shallow_water_fluxes, HIP_SYMBOL(d_shallow_water_fluxes),
                         sizeof(flux_func_ptr));

    hipError_t err;

    // if first layer is single layer SWE, need to restrict multilayer SWE
    // data (where initial data has been defined) to this

    // need some kind of logic here to define where need to do restriction/prolonging at the start
    bool do_restrict_and_prolong = true;
    if (tstart == 0 && do_restrict_and_prolong) {
        for (int i = min(c_in,nlevels-1); i > 0; i--) {
            // TODO: check if need to do BCS stuff here
            hipMemcpy(U_d, Us_h[i-1],
                nxs[i-1]*nys[i-1]*nzs[i-1]*vec_dims[i-1]*sizeof(float),
                hipMemcpyHostToDevice);
            hipMemcpy(Up_d, Us_h[i],
                nxs[i]*nys[i]*nzs[i]*vec_dims[i]*sizeof(float),
                hipMemcpyHostToDevice);
            // select restriction algorithm
            if (models[i-1] == 'M' && models[i] == 'C') {
              // compressible to multilayer SWE
              restrict_comp_to_swe(kernels, threads, blocks,
                        cumulative_kernels,
                        U_d, Up_d, nxs_d, nys_d, nzs_d,
                        dz/pow(r, i), zmin, matching_indices_d,
                        rho_d, gamma, ng, rank, qf_swe, i-1, p_const_d, R, alpha0);
            } else if (models[i-1] == 'M' && models[i] == 'M') {
              // multilayer SWE to multilayer SWE
              restrict_multiswe_to_multiswe(kernels, threads, blocks,
                        cumulative_kernels,
                        U_d, Up_d, nxs_d, nys_d, nzs_d,
                        matching_indices_d,
                        ng, rank, i-1);
            } else if (models[i-1] == 'C' && models[i] == 'C') {
              // compressible to compressible
              restrict_comp_to_comp(kernels, threads, blocks,
                        cumulative_kernels,
                        U_d, Up_d, nxs_d, nys_d, nzs_d,
                        matching_indices_d,
                        ng, rank, i-1);
            } else if (models[i-1] == 'S' && (models[i] == 'S' || models[i] == 'M')) {
              // multilayer SWE to single layer SWE
              restrict_swe_to_swe(kernels, threads, blocks,
                        cumulative_kernels,
                        U_d, Up_d, nxs_d, nys_d, nzs_d,
                        matching_indices_d,
                        ng, rank, i-1);
            }
            hipMemcpy(Us_h[i-1], U_d,
                  nxs[i-1]*nys[i-1]*nzs[i-1]*vec_dims[i-1]*sizeof(float),
                  hipMemcpyDeviceToHost);

            // enforce boundaries
            for (int x = 0; x < matching_indices[(i-1)*4]+ng; x++) {
                for (int y = 0; y < nys[i-1]*nzs[i-1]; y++) {
                    for (int n = 0; n < vec_dims[i-1]; n++) {
                        Us_h[i-1][(y * nxs[i-1] + x) * vec_dims[i-1] + n] =
                            Us_h[i-1][(y * nxs[i-1] + matching_indices[(i-1)*4]+ng) * vec_dims[i-1] + n];
                    }
                }
            }
            for (int x = matching_indices[(i-1)*4+1]; x < nxs[i-1]; x++) {
                for (int y = 0; y < nys[i-1]*nzs[i-1]; y++) {
                    for (int n = 0; n < vec_dims[i-1]; n++) {
                        Us_h[i-1][(y * nxs[i-1] + x) * vec_dims[i-1] + n] =
                            Us_h[i-1][(y * nxs[i-1] + matching_indices[(i-1)*4+1]-ng) * vec_dims[i-1] + n];
                    }
                }
            }
            for (int z = 0; z < nzs[i-1]; z++) {
                for (int x = 0; x < nxs[i-1]; x++) {
                    for (int y = 0; y < matching_indices[(i-1)*4+2]+ng; y++) {
                        for (int n = 0; n < vec_dims[i-1]; n++) {
                            Us_h[i-1][((z * nys[i-1] + y) * nxs[i-1] + x) * vec_dims[i-1] + n] =
                                Us_h[i-1][((z * nys[i-1] + matching_indices[(i-1)*4+2]+ng) * nxs[i-1] + x) * vec_dims[i-1] + n];
                        }
                    }
                }
                for (int x = 0; x < nxs[i-1]; x++) {
                    for (int y = matching_indices[(i-1)*4+3]; y < nys[i-1]; y++) {
                        for (int n = 0; n < vec_dims[i-1]; n++) {
                            Us_h[i-1][((z * nys[i-1] + y) * nxs[i-1] + x) * vec_dims[i-1] + n] =
                                Us_h[i-1][((z * nys[i-1] + matching_indices[(i-1)*4+3]-ng) * nxs[i-1] + x) * vec_dims[i-1] + n];
                        }
                    }
                }
            }
        }

        //for (int i = 0; i < nxs[0]*nys[0]*nzs[0]; i++) {
            //cout << Us_h[0][i*4+1] << ' ' << Us_h[0][i*4+1] << '\n';
        //}

        // prolong data from coarser to finer grids
        for (int i = c_in; i < (nlevels-1); i++) {
            hipMemcpy(U_d, Us_h[i],
                    nxs[i]*nys[i]*nzs[i]*vec_dims[i]*sizeof(float),
                    hipMemcpyHostToDevice);
            hipMemcpy(Up_d, Us_h[i+1],
                    nxs[i+1]*nys[i+1]*nzs[i+1]*vec_dims[i+1]*sizeof(float),
                    hipMemcpyHostToDevice);

            // select prolongation algorithm
            if (models[i] == 'M' && models[i+1] == 'C') {
                // multilayer SWE to compressible
                for (int j = 0; j < nxs[i]*nys[i]*nzs[i]; j++) {
                    pphi[j] = Us_h[i][j*4];
                }
                hipMemcpy(old_phi_d, pphi,
                        nxs[i]*nys[i]*nzs[i]*sizeof(float),
                        hipMemcpyHostToDevice);
                prolong_swe_to_comp(kernels, threads, blocks,
                             cumulative_kernels,
                             U_d, Up_d, nxs_d, nys_d, nzs_d,
                             dz/pow(r, i), dt/pow(r, i), zmin,
                             rho_d, gamma, matching_indices_d, ng, rank,
                             q_comp_d, old_phi_d, i, false, R);
            } else if (models[i] == 'M' && models[i+1] == 'M') {
                // multilayer SWE to multilayer SWE
                prolong_multiswe_to_multiswe(kernels, threads, blocks,
                             cumulative_kernels,
                             U_d, Up_d, nxs_d, nys_d, nzs_d,
                             matching_indices_d, ng, rank, i, false);
            } else if (models[i] == 'C' && models[i+1] == 'C') {
                // compressible to compressible
                prolong_comp_to_comp(kernels, threads, blocks,
                             cumulative_kernels,
                             U_d, Up_d, nxs_d, nys_d, nzs_d,
                             matching_indices_d, ng, rank, i, false);
            } else if (models[i] == 'S' && (models[i+1] == 'S' || models[i+1] == 'M')) {
                // single layer SWE to multilayer SWE
                prolong_swe_to_swe(kernels, threads, blocks,
                             cumulative_kernels,
                             U_d, Up_d, nxs_d, nys_d, nzs_d,
                             matching_indices_d, ng, rank, i, false);
            }

            err = hipGetLastError();
            if (err != hipSuccess){
                cout << "After prolonging\n";
                printf("Error: %s\n", hipGetErrorString(err));
            }

            hipMemcpy(Us_h[i+1], Up_d,
                    nxs[i+1]*nys[i+1]*nzs[i+1]*vec_dims[i+1]*sizeof(float),
                    hipMemcpyDeviceToHost);

            err = hipGetLastError();
            if (err != hipSuccess){
                cout << "After copying\n";
                printf("Error: %s\n", hipGetErrorString(err));
            }

            bool do_z = true;
            if (models[i+1] == 'M' || models[i+1] == 'S') {
                do_z = false;
            }

            // enforce boundaries
            if (n_processes == 1) {
                bcs_fv(Us_h[i+1], nxs[i+1], nys[i+1], nzs[i+1],
                        ng, vec_dims[i+1], false, do_z);
                if (do_z) {
                    enforce_hse(Us_h[i+1], Us_h[m_in],
                                    nxs, nys, nzs, ng,
                                    i+1, m_in, zmin, dz/pow(r, i+1),
                                    matching_indices, gamma, R, alpha0);
                }

            } else {
                int y_size = kernels[0].y*blocks[0].y*threads[0].y - 2*ng;
                bcs_mpi(Us_h[i+1], nxs[i+1], nys[i+1], nzs[i+1],
                        vec_dims[i+1], ng, comm, status, rank,
                        n_processes, y_size, do_z, false);
            }
        }
        // NOTE: Initial conditions for multiscale test
        /*if (models[nlevels-1] == 'C') { // there's at least one compressible level
            for (int z = 0; z < nzs[nlevels-1]; z++) {
                for (int y = 0; y < nys[nlevels-1]; y++) {
                    for (int x = 0; x < nxs[nlevels-1]; x++) {
                        float max_v = 0.3;
                        float r = sqrt(
                            (x - 0.5*nxs[nlevels-1])*(x - 0.5*nxs[nlevels-1]) +
                            (y - 0.5*nys[nlevels-1])*(y - 0.5*nys[nlevels-1]));
                        float v = 0.0;
                        if (r < 0.05 * nxs[nlevels-1]) {
                            v = 20.0 * max_v * r / nxs[nlevels-1];
                        } else if (r < 0.1 * nxs[nlevels-1]) {
                            v = 2.0 * 20.0 * max_v * 0.05 - 20.0 * max_v * r / nxs[nlevels-1];
                        }
                        float D = Us_h[nlevels-1][((z*nys[nlevels-1] + y) * nxs[nlevels-1] + x) * vec_dims[nlevels-1]];

                        if (r > 0.0) {
                            // Sx
                            Us_h[nlevels-1][((z * nys[nlevels-1] + y) * nxs[nlevels-1] + x) * vec_dims[nlevels-1] + 1]
                                = - D * v * (y - 0.5*nys[nlevels-1]) / r;
                            Us_h[nlevels-1][((z * nys[nlevels-1] + y) * nxs[nlevels-1] + x) * vec_dims[nlevels-1] + 2]
                                = D * v * (x - 0.5*nxs[nlevels-1]) / r;
                        }
                    }
                }
            }
        }*/
    }

    hid_t outFile;
    hid_t * dset = new hid_t[n_print_levels];
    hid_t * mem_space = new hid_t[n_print_levels];
    hid_t * file_space = new hid_t[n_print_levels];

    if (rank == 0) {
        initialise_hdf5_file(filename, nt, dprint,
            nzs, nys, nxs, vec_dims, n_print_levels,
            print_levels, Us_h, &outFile, dset, mem_space, file_space,
            param_filename);
    }

    err = hipGetLastError();
    if (err != hipSuccess){
        cout << "Before evolution\n";
        printf("Error: %s\n", hipGetErrorString(err));
    }

    // main loop
    for (int t = tstart; t < nt; t++) {
        cout << "Evolving t = " << t << '\n';

        // Do evolutions on grids
        for (int i = (nlevels-1); i >= 0; i--) {

            flux_func_ptr flux_func = h_compressible_fluxes;
            // HACK - set back to true
            bool do_z = true;
            if (models[i] == 'M' || models[i] == 'S') { // SWE
                flux_func = h_shallow_water_fluxes;
                do_z = false;
            }

            for (int j = 0; j < pow(r, i); j++) {
                hipMemcpy(U_d, Us_h[i],
                        nxs[i]*nys[i]*nzs[i]*vec_dims[i]*sizeof(float),
                        hipMemcpyHostToDevice);
                // TODO: fix dz calculation (i.e. need to work out how to store it in a way such that the compressible grids are getting the correct value)

                err = hipGetLastError();
                if (err != hipSuccess) {
                    cout << "Before rk3\n";
                    printf("Error: %s\n", hipGetErrorString(err));
                }

                rk3(kernels, threads, blocks, cumulative_kernels,
                        U_d, U_half_d, Up_d,
                        qx_p_d, qx_m_d, qy_p_d, qy_m_d, qz_p_d, qz_m_d,
                        fx_p_d, fx_m_d, fy_p_d, fy_m_d, fz_p_d, fz_m_d,
                        i,
                        nxs, nys, nzs, vec_dims, ng,
                        alpha0, R, gamma,
                        dx/pow(r, i), dy/pow(r, i), dz/pow(r, i),
                        dt/pow(r, i),
                        Up_h, F_h, Us_h[i],
                        comm, status, rank, n_processes,
                        flux_func, do_z, (i==0) ? periodic : false,
                        m_in, Us_h[m_in], matching_indices, zmin);

                hipDeviceSynchronize();

                // add gravitational source terms
                if (models[i] == 'C') {

                    hipMemcpy(U_d, Us_h[i],
                            nxs[i]*nys[i]*nzs[i]*vec_dims[i]*sizeof(float),
                            hipMemcpyHostToDevice);

                    int kx_offset = 0;
                    int ky_offset = (kernels[0].y * blocks[0].y *
                                 threads[0].y - 2*ng) * rank;

                    for (int l = 0; l < kernels[rank].y; l++) {
                        kx_offset = 0;
                        for (int k = 0; k < kernels[rank].x; k++) {
                            grav_sources<<<blocks[k_offset + l * kernels[rank].x + k], threads[k_offset + l * kernels[rank].x + k]>>>(
                                   U_d, gamma,
                                   nxs[i], nys[i], nzs[i], vec_dims[i],
                                   zmin, R, alpha0,
                                   dz/pow(r, i), dt/pow(r, i),
                                   kx_offset, ky_offset);
                            kx_offset += blocks[k_offset + l *
                                kernels[rank].x + k].x *
                                threads[k_offset+l*kernels[rank].x+k].x -
                                2*ng;
                        }
                        ky_offset += blocks[k_offset + l *
                            kernels[rank].x].y *
                            threads[k_offset + l * kernels[rank].x].y -
                            2*ng;
                    }

                    hipMemcpy(Us_h[i], U_d,
                            nxs[i]*nys[i]*nzs[i]*vec_dims[i]*sizeof(float),
                            hipMemcpyDeviceToHost);
                }

                /*if (models[i] == 'C') {
                    // hack on the burning
                    float * H = new float[nxs[i]*nys[i]*nzs[i]];
                    calc_Q(rho, Us_h[i], nxs[i], nys[i], nzs[i], gamma,
                           H, Cv, gamma_up);
                    for (int z = 0; z < nzs[i]; z++) {
                        for (int y = ng; y < nys[i]-ng; y++) {
                            for (int x = ng; x < nxs[i] - ng; x++) {
                                // tau
                                Us_h[i][((z*nys[i]+y)*nxs[i]+x)*6 + 4] +=
                                    dt/pow(r, i) * 0.5 * alpha *
                                    Us_h[i][((z*nys[i]+y)*nxs[i]+x)*6] *
                                    H[(z * nys[i] + y) * nxs[i] + x];
                                float X_dot =
                                    H[(z*nys[i] + y)*nxs[i] + x] / E_He;
                                // DX
                                Us_h[i][((z*nys[i]+y)*nxs[i]+x)*6+5] +=
                                    dt/pow(r, i) * 0.5 * alpha * rho[0] *
                                    X_dot;
                            }
                        }
                    }
                    delete[] H;
                } else if (models[i] == 'M') { // SWE burning
                    // update old_phi
                    for (int j = 0; j < nxs[i]*nys[i]*nzs[i]; j++) {
                        pphi[j] = Us_h[i][j*4];
                    }
                    hipMemcpy(old_phi_d, pphi,
                            nxs[i]*nys[i]*nzs[i]*sizeof(float),
                            hipMemcpyHostToDevice);

                    hipMemcpy(Up_d, Us_h[i],
                            nxs[i]*nys[i]*nzs[i]*4*sizeof(float),
                            hipMemcpyHostToDevice);
                    hipMemcpy(U_half_d, Us_h[i],
                            nxs[i]*nys[i]*nzs[i]*4*sizeof(float),
                            hipMemcpyHostToDevice);

                    int kx_offset = 0;
                    ky_offset = (kernels[0].y * blocks[0].y *
                                 threads[0].y - 2*ng) * rank;

                    for (int l = 0; l < kernels[rank].y; l++) {
                        kx_offset = 0;
                        for (int k = 0; k < kernels[rank].x; k++) {
                            evolve_fv_heating<<<blocks[k_offset + l * kernels[rank].x + k], threads[k_offset + l * kernels[rank].x + k]>>>(
                                   Up_d, U_half_d,
                                   qx_p_d, qx_m_d, qy_p_d, qy_m_d,
                                   fx_p_d, fx_m_d, fy_p_d, fy_m_d,
                                   sum_phs_d, rho_d, Q_d,
                                   nxs[i], nys[i], nzs[i], alpha, gamma,
                                   dx/pow(r, i), dy/pow(r, i),
                                   dt/pow(r, i),
                                   burning, Cv, E_He,
                                   kx_offset, ky_offset);
                            kx_offset += blocks[k_offset + l *
                                kernels[rank].x + k].x *
                                threads[k_offset+l*kernels[rank].x+k].x -
                                2*ng;
                        }
                        ky_offset += blocks[k_offset + l *
                            kernels[rank].x].y *
                            threads[k_offset + l * kernels[rank].x].y -
                            2*ng;
                    }

                    hipMemcpy(Up_h, Up_d,
                            nxs[i]*nys[i]*nzs[i]*4*sizeof(float),
                            hipMemcpyDeviceToHost);
                    hipMemcpy(sum_phs_h, sum_phs_d,
                            nxs[i]*nys[i]*nzs[i]*sizeof(float),
                            hipMemcpyDeviceToHost);

                    // enforce boundaries
                    if (n_processes == 1) {
                        bcs_fv(Up_h, nxs[i], nys[i], nzs[i], ng, 4, (i==0) ? periodic : false, do_z);
                        bcs_fv(sum_phs_h, nxs[i], nys[i], nzs[i], ng, 1,
                               (i==0) ? periodic : false, do_z);
                    } else {
                        int y_size = kernels[0].y * blocks[0].y *
                                     threads[0].y - 2*ng;
                        bcs_mpi(Up_h, nxs[i], nys[i], nzs[i], 4, ng, comm,
                                status, rank, n_processes, y_size, false,
                                (i==0) ? periodic : false);
                        bcs_mpi(sum_phs_h, nxs[i], nys[i], nzs[i], 1, ng,
                                comm, status, rank, n_processes, y_size,
                                false, (i==0) ? periodic : false);
                    }

                    hipMemcpy(Up_d, Up_h,
                               nxs[i]*nys[i]*nzs[i]*4*sizeof(float),
                               hipMemcpyHostToDevice);
                    hipMemcpy(sum_phs_d, sum_phs_h,
                               nxs[i]*nys[i]*nzs[i]*sizeof(float),
                               hipMemcpyHostToDevice);

                    kx_offset = 0;
                    ky_offset = (kernels[0].y * blocks[0].y *
                                 threads[0].y - 2*ng) * rank;

                    for (int p = 0; p < kernels[rank].y; p++) {
                        kx_offset = 0;
                        for (int q = 0; q < kernels[rank].x; q++) {
                            evolve2<<<blocks[k_offset + p * kernels[rank].x + q], threads[k_offset + p * kernels[rank].x + q]>>>(U_d,
                                   Up_d, U_half_d, sum_phs_d,
                                   nxs[i], nys[i], nzs[i], ng, alpha,
                                   dx/pow(r, i), dy/pow(r, i), dt/pow(r, i),
                                   kx_offset, ky_offset);
                            kx_offset +=
                                blocks[k_offset+p*kernels[rank].x + q].x *
                                threads[k_offset+p*kernels[rank].x+q].x -
                                2*ng;
                        }
                        ky_offset += blocks[k_offset + p *
                            kernels[rank].x].y *
                            threads[k_offset + p * kernels[rank].x].y -
                            2*ng;
                    }

                    hipDeviceSynchronize();

                    err = hipGetLastError();

                    if (err != hipSuccess)
                        printf("Error: %s\n", hipGetErrorString(err));

                    // boundaries
                    hipMemcpy(Us_h[i], U_d,
                               nxs[i]*nys[i]*nzs[i]*4*sizeof(float),
                               hipMemcpyDeviceToHost);
                    if (n_processes == 1) {
                        bcs_fv(Us_h[i], nxs[i], nys[i], nzs[i], ng, 4,
                               (i==0) ? periodic : false, do_z);
                    } else {
                        int y_size = kernels[0].y * blocks[0].y *
                                     threads[0].y - 2*ng;
                        bcs_mpi(Us_h[i], nxs[i], nys[i], nzs[i], 4, ng,
                                comm, status, rank, n_processes, y_size,
                                false, (i==0) ? periodic : false);
                    }
                    hipMemcpy(U_d, Us_h[i],
                               nxs[i]*nys[i]*nzs[i]*4*sizeof(float),
                               hipMemcpyHostToDevice);
                }*/

                if (n_processes == 1) {
                    bcs_fv(Us_h[i], nxs[i], nys[i], nzs[i], ng,
                           vec_dims[i], (i==0) ? periodic : false, do_z);
                    if (do_z) {
                        enforce_hse(Us_h[i], Us_h[m_in],
                                       nxs, nys, nzs, ng,
                                       i, m_in, zmin, dz/pow(r, i),
                                       matching_indices, gamma, R, alpha0);
                    }
                } else {
                    int y_size = kernels[0].y * blocks[0].y *
                                 threads[0].y - 2*ng;
                    bcs_mpi(Us_h[i], nxs[i], nys[i], nzs[i], vec_dims[i],
                            ng, comm, status, rank, n_processes, y_size,
                            false, (i==0) ? periodic : false);
                }
            }
        }

        //for (int i = 0; i < nxs[0]*nys[0]*nzs[0]; i++) {
            //cout << Us_h[1][i*6] << '\n';
        //}

        for (int i = (nlevels-1); i > 0; i--) {
            // restrict to coarse grid
            // copy to device
            hipMemcpy(Up_d, Us_h[i-1],
                nxs[i-1]*nys[i-1]*nzs[i-1]*vec_dims[i-1]*sizeof(float),
                hipMemcpyHostToDevice);
            hipMemcpy(U_d, Us_h[i],
                nxs[i]*nys[i]*nzs[i]*vec_dims[i]*sizeof(float),
                hipMemcpyHostToDevice);
            // select restriction algorithm
            if (models[i-1] == 'M' && models[i] == 'C') {
                // compressible to multilayer SWE
                restrict_comp_to_swe(kernels, threads, blocks,
                          cumulative_kernels,
                          Up_d, U_d, nxs_d, nys_d, nzs_d,
                          dz/pow(r, i), zmin, matching_indices_d,
                          rho_d, gamma, ng, rank, qf_swe, i-1, p_const_d,
                          R, alpha0);
            } else if (models[i-1] == 'M' && models[i] == 'M') {
                // multilayer SWE to multilayer SWE
                restrict_multiswe_to_multiswe(kernels, threads, blocks,
                          cumulative_kernels,
                          Up_d, U_d, nxs_d, nys_d, nzs_d,
                          matching_indices_d,
                          ng, rank, i-1);
            } else if (models[i-1] == 'C' && models[i] == 'C') {
                // compressible to compressible
                restrict_comp_to_comp(kernels, threads, blocks,
                          cumulative_kernels,
                          Up_d, U_d, nxs_d, nys_d, nzs_d,
                          matching_indices_d,
                          ng, rank, i-1);
            } else if (models[i-1] == 'S' && (models[i] == 'S' || models[i] == 'M')) {
                // multilayer SWE to single layer SWE
                restrict_swe_to_swe(kernels, threads, blocks,
                          cumulative_kernels,
                          Up_d, U_d, nxs_d, nys_d, nzs_d,
                          matching_indices_d,
                          ng, rank, i-1);
            }
            err = hipGetLastError();
            if (err != hipSuccess){
                cout << "After restricting\n";
                printf("Error: %s\n", hipGetErrorString(err));
            }

            hipMemcpy(Us_h[i-1], Up_d,
                nxs[i-1]*nys[i-1]*nzs[i-1]*vec_dims[i-1]*sizeof(float),
                hipMemcpyDeviceToHost);

            err = hipGetLastError();
            if (err != hipSuccess){
                cout << "After copying\n";
                printf("Error: %s\n", hipGetErrorString(err));
            }
        }

        /*for (int i = 0; i < nxs[0]*nys[0]*nzs[0]; i++) {
            cout << Us_h[0][i*4] << '\n';
        }*/

        // prolong data down from coarse grids to fine grids
        for (int i = 0; i < (nlevels-1); i++) {
            hipMemcpy(U_d, Us_h[i],
                    nxs[i]*nys[i]*nzs[i]*vec_dims[i]*sizeof(float),
                    hipMemcpyHostToDevice);
            hipMemcpy(Up_d, Us_h[i+1],
                    nxs[i+1]*nys[i+1]*nzs[i+1]*vec_dims[i+1]*sizeof(float),
                    hipMemcpyHostToDevice);

            // select prolongation algorithm
            if (models[i] == 'M' && models[i+1] == 'C') {
                // multilayer SWE to compressible
                for (int j = 0; j < nxs[i]*nys[i]*nzs[i]; j++) {
                    pphi[j] = Us_h[i][j*4];
                }
                hipMemcpy(old_phi_d, pphi,
                        nxs[i]*nys[i]*nzs[i]*sizeof(float),
                        hipMemcpyHostToDevice);
                prolong_swe_to_comp(kernels, threads, blocks,
                             cumulative_kernels,
                             U_d, Up_d, nxs_d, nys_d, nzs_d,
                             dz/pow(r, i), dt/pow(r, i), zmin,
                             rho_d, gamma, matching_indices_d, 2*ng, rank,
                             q_comp_d, old_phi_d, i, true, R);
            } else if (models[i] == 'M' && models[i+1] == 'M') {
                // multilayer SWE to multilayer SWE
                prolong_multiswe_to_multiswe(kernels, threads, blocks,
                             cumulative_kernels,
                             U_d, Up_d, nxs_d, nys_d, nzs_d,
                             matching_indices_d, 2*ng, rank, i, true);
            } else if (models[i] == 'C' && models[i+1] == 'C') {
                // compressible to compressible
                prolong_comp_to_comp(kernels, threads, blocks,
                             cumulative_kernels,
                             U_d, Up_d, nxs_d, nys_d, nzs_d,
                             matching_indices_d, ng, rank, i, true);
            } else if (models[i] == 'S' && (models[i+1] == 'S' || models[i+1] == 'M')) {
                // single layer SWE to multilayer SWE
                prolong_swe_to_swe(kernels, threads, blocks,
                             cumulative_kernels,
                             U_d, Up_d, nxs_d, nys_d, nzs_d,
                             matching_indices_d, 2*ng, rank, i, true);
            }

            err = hipGetLastError();
            if (err != hipSuccess){
                cout << "After prolonging\n";
                printf("Error: %s\n", hipGetErrorString(err));
            }

            hipMemcpy(Us_h[i+1], Up_d,
                    nxs[i+1]*nys[i+1]*nzs[i+1]*vec_dims[i+1]*sizeof(float),
                    hipMemcpyDeviceToHost);

            err = hipGetLastError();
            if (err != hipSuccess){
                cout << "After copying\n";
                printf("Error: %s\n", hipGetErrorString(err));
            }

        }

        if ((t+1) % dprint == 0) {
            for (int i = 0; i < n_print_levels; i++) {
                print_timestep(rank, n_processes, print_levels[i],
                               nxs, nys, nzs, vec_dims, ng, t, comm, status,
                               kernels, threads, blocks, Us_h,
                               dset[i], mem_space[i], file_space[i], dprint);
            }
        }
    }

    if (rank == 0) {
        close_hdf5_file(n_print_levels, mem_space, outFile);
    }

    // delete some stuff
    hipFree(rho_d);
    hipFree(Q_d);
    hipFree(old_phi_d);
    hipFree(sum_phs_d);

    hipFree(U_d);
    hipFree(Up_d);
    hipFree(U_half_d);
    hipFree(F_d);

    hipFree(nxs_d);
    hipFree(nys_d);
    hipFree(nzs_d);

    hipFree(qx_p_d);
    hipFree(qx_m_d);
    hipFree(qy_p_d);
    hipFree(qy_m_d);
    hipFree(qz_p_d);
    hipFree(qz_m_d);
    hipFree(fx_p_d);
    hipFree(fx_m_d);
    hipFree(fy_p_d);
    hipFree(fy_m_d);
    hipFree(fz_p_d);
    hipFree(fz_m_d);
    hipFree(q_comp_d);
    hipFree(qf_swe);
    hipFree(matching_indices_d);

    delete[] kernels;
    delete[] cumulative_kernels;
    delete[] threads;
    delete[] blocks;
    delete[] pphi;
    delete[] sum_phs_h;

    delete[] U_h;
    delete[] Up_h;
    delete[] F_h;

    delete[] dset;
    delete[] mem_space;
    delete[] file_space;
}
