#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <helper_functions.h>
#include "../Mesh_cuda.h"
#include "../mesh_cuda_kernel.h"
#include "unit_tests.h"

using namespace std;

/*
This does some unit tests on some of the functions in swerve.
*/

bool test_cons_to_prim_comp_d_wrapper() {
    int ntests = 100;
    bool *passed_vec_d, *passed_vec;
    passed_vec = (bool *)malloc(ntests*sizeof(bool));
    hipMalloc((void**)&passed_vec_d, ntests*sizeof(bool));

    float * q_prim, *q_prim_d;
    q_prim = (float *)malloc(5*ntests*sizeof(float));
    hipMalloc((void**)&q_prim_d, 5*ntests*sizeof(float));

    for (int i = 0; i < ntests; i++) {
        q_prim[i*5+0] = 10*r();
        q_prim[i*5+1] = 0.8*r()-0.4;
        q_prim[i*5+2] = r()-0.5;
        q_prim[i*5+3] = r()-0.5;
        q_prim[i*5+4] = 15*r();
    }
    hipMemcpy(q_prim_d, q_prim, ntests*5*sizeof(float), hipMemcpyHostToDevice);
    test_cons_to_prim_comp_d<<<1,ntests>>>(passed_vec_d, q_prim_d);
    hipMemcpy(passed_vec, passed_vec_d, ntests*sizeof(bool), hipMemcpyDeviceToHost);

    bool passed = true;
    for (int i = 0; i < ntests; i++) {
        if (!(passed_vec[i])) {
            passed = false;
        }
    }

    free(passed_vec);
    hipFree(passed_vec_d);
    hipFree(q_prim_d);
    free(q_prim);

    return passed;
}

bool test_swe_from_compressible() {
    bool passed = true;
    float gamma_up[] = {0.80999862,  0.0 ,  0.0,  0.0,  0.80999862,
        0.0,  0.0,  0.0,  0.80999862};
    float gamma = 5.0/3.0;
    int kx_offset = 0;
    int ky_offset = 0;
    float p_floor = 1.0;
    const int nxf = 5;
    const int nyf = 1;
    const int nz = 2;

    float * gamma_up_d;
    hipMalloc((void**)&gamma_up_d, 9*sizeof(float));
    hipMemcpy(gamma_up_d, gamma_up, 9*sizeof(float), hipMemcpyHostToDevice);

    float q[] = {1.0,0.0,0.0,0.0,1.0,
                 0.001, 0.0, 0.0, 0.0, 0.001,
                 1000., 0.0, 0.0, 0.0, 1000.,
                 1.05245657, 0.59075458, 0.59075458, 0.59075458, 1.23464966,
                 1.03406473, 0.2142144, 0.2142144, 0.0, 0.03634062, 1.0,0.0,0.0,0.0,1.0,
                  0.001, 0.0, 0.0, 0.0, 0.001,
                  1000., 0.0, 0.0, 0.0, 1000.,
                  1.05245657, 0.59075458, 0.59075458, 0.59075458, 1.23464966,
                  1.03406473, 0.2142144, 0.2142144, 0.0, 0.03634062};
    float q_swe[] = {0.39233170120469052, 0.0, -0.0,
                     -2.3707704103881642, -0.0, 0.0,
                     3.1554338127975452, 0.0, -0.0,
                     0.41291207788984285, 0.086914406123075383, 0.086914406123075383,
                     0.00068880264357815526, 0.000142453303890117, 0.000142453303890117};
    float rho[] = {1.0, 1.0e-3, 1.0e3, 1.0, 1.0,1.0, 1.0e-3, 1.0e3, 1.0, 1.0};

    float * q_d, *q_swe_new, * q_swe_d, *rho_d;
    hipMalloc((void**)&q_d, 5*nxf*nz*sizeof(float));
    q_swe_new = (float *)malloc(3*nxf*nz*sizeof(float));
    hipMalloc((void**)&q_swe_d, 3*nxf*nz*sizeof(float));
    hipMalloc((void**)&rho_d, nxf*nz*sizeof(float));
    hipMemcpy(q_d, q, 5*nxf*nz*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(rho_d, rho, nxf*nz*sizeof(float), hipMemcpyHostToDevice);

    dim3 block(nxf, nyf, nz);

    swe_from_compressible<<<1,block>>>(q_d, q_swe_d, nxf, nyf, nz, gamma_up_d,
                                   rho_d, gamma, kx_offset, ky_offset,
                                   p_floor);

    hipMemcpy(q_swe_new, q_swe_d, 3*nxf*nz*sizeof(float), hipMemcpyDeviceToHost);

    const float tol = 1.0e-5;
    for (int i = 0; i < nxf; i++) {
        for (int n = 0; n < 3; n++) {
            if ((abs((q_swe[i*3+n] - q_swe_new[i*3+n]) / q_swe[i*3+n]) > tol) && (abs(q_swe[i*3+n] - q_swe_new[i*3+n]) > 0.1*tol)) {
                printf("%f, %f\n", q_swe[i*3+n], q_swe_new[i*3+n]);
                passed = false;
            }
        }
    }

    hipFree(gamma_up_d);
    hipFree(q_d);
    free(q_swe_new);
    hipFree(q_swe_d);
    hipFree(rho_d);

    return passed;
}

void run_cuda_tests() {

    bool passed = true;
    bool *passed_d;
    hipMalloc((void**)&passed_d, sizeof(bool));

    test_find_height<<<1,1>>>(passed_d);
    hipMemcpy(&passed, passed_d, sizeof(bool), hipMemcpyDeviceToHost);
    if (passed) {
        cout << "find_height passed!\n";
    } else {
        cout << "find_height did not pass :(\n";
    }

    test_find_pot<<<1,1>>>(passed_d);
    hipMemcpy(&passed, passed_d, sizeof(bool), hipMemcpyDeviceToHost);
    if (passed) {
        cout << "find_pot passed!\n";
    } else {
        cout << "find_pot did not pass :(\n";
    }

    test_rhoh_from_p<<<1,1>>>(passed_d);
    hipMemcpy(&passed, passed_d, sizeof(bool), hipMemcpyDeviceToHost);
    if (passed) {
        cout << "rhoh_from_p passed!\n";
    } else {
        cout << "rhoh_from_p did not pass :(\n";
    }

    test_p_from_rhoh<<<1,1>>>(passed_d);
    hipMemcpy(&passed, passed_d, sizeof(bool), hipMemcpyDeviceToHost);
    if (passed) {
        cout << "p_from_rhoh passed!\n";
    } else {
        cout << "p_from_rhoh did not pass :(\n";
    }

    test_p_from_rho_eps<<<1,1>>>(passed_d);
    hipMemcpy(&passed, passed_d, sizeof(bool), hipMemcpyDeviceToHost);
    if (passed) {
        cout << "p_from_rho_eps passed!\n";
    } else {
        cout << "p_from_rho_eps did not pass :(\n";
    }

    test_hdot<<<1,1>>>(passed_d);
    hipMemcpy(&passed, passed_d, sizeof(bool), hipMemcpyDeviceToHost);
    if (passed) {
        cout << "h_dot passed!\n";
    } else {
        cout << "h_dot did not pass :(\n";
    }

    test_calc_As<<<1,1>>>(passed_d);
    hipMemcpy(&passed, passed_d, sizeof(bool), hipMemcpyDeviceToHost);
    if (passed) {
        cout << "calc_As passed!\n";
    } else {
        cout << "calc_As did not pass :(\n";
    }

    passed = test_cons_to_prim_comp_d_wrapper();
    if (passed) {
        cout << "cons_to_prim_comp_d passed!\n";
    } else {
        cout << "cons_to_prim_comp_d did not pass :(\n";
    }

    test_shallow_water_fluxes<<<1,1>>>(passed_d);
    hipMemcpy(&passed, passed_d, sizeof(bool), hipMemcpyDeviceToHost);
    if (passed) {
        cout << "shallow_water_fluxes passed!\n";
    } else {
        cout << "shallow_water_fluxes did not pass :(\n";
    }

    test_compressible_fluxes<<<1,1>>>(passed_d);
    hipMemcpy(&passed, passed_d, sizeof(bool), hipMemcpyDeviceToHost);
    if (passed) {
        cout << "compressible_fluxes passed!\n";
    } else {
        cout << "compressible_fluxes did not pass :(\n";
    }

    test_p_from_swe<<<1,1>>>(passed_d);
    hipMemcpy(&passed, passed_d, sizeof(bool), hipMemcpyDeviceToHost);
    if (passed) {
        cout << "p_from_swe passed!\n";
    } else {
        cout << "p_from_swe did not pass :(\n";
    }

    passed = test_swe_from_compressible();
    if (passed) {
        cout << "swe_from_compressible passed!\n";
    } else {
        cout << "swe_from_compressible did not pass :(\n";
    }

    test_height_err<<<1,1>>>(passed_d);
    hipMemcpy(&passed, passed_d, sizeof(bool), hipMemcpyDeviceToHost);
    if (passed) {
        cout << "height_err passed!\n";
    } else {
        cout << "height_err did not pass :(\n";
    }

    hipFree(passed_d);
}
