#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <helper_functions.h>
#include "../Mesh_cuda.h"
#include "../mesh_cuda_kernel.h"
#include "unit_tests.h"

using namespace std;

/*
This does some unit tests on some of the functions in swerve.
*/


/*__global__ void test_find_height(bool * passed) {

    float ph[] = {1.0e-3, 1.0, 1.0e3};
    float h[] = {10001.000333, 2.313035285, 1.0};

    const float tol = 1.0e-5;

    for (int i = 0; i < 3; i++) {
        if ((abs((h[i] - find_height(ph[i])) / h[i]) > tol) && (abs(h[i] - find_height(ph[i])) > 0.01*tol)) {
            printf("%f, %f\n", h[i], find_height(ph[i]));
            *passed = false;
        }
    }

    *passed = true;
}*/

void run_cuda_tests() {

    bool passed = true;
    bool *passed_d;
    hipMalloc((void**)&passed_d, sizeof(bool));

    test_find_height<<<1,1>>>(passed_d);
    hipMemcpy(&passed, passed_d, sizeof(bool), hipMemcpyDeviceToHost);
    if (passed) {
        cout << "find_height passed!\n";
    } else {
        cout << "find_height did not pass :(\n";
    }

    test_find_pot<<<1,1>>>(passed_d);
    hipMemcpy(&passed, passed_d, sizeof(bool), hipMemcpyDeviceToHost);
    if (passed) {
        cout << "find_pot passed!\n";
    } else {
        cout << "find_pot did not pass :(\n";
    }

    test_rhoh_from_p<<<1,1>>>(passed_d);
    hipMemcpy(&passed, passed_d, sizeof(bool), hipMemcpyDeviceToHost);
    if (passed) {
        cout << "rhoh_from_p passed!\n";
    } else {
        cout << "rhoh_from_p did not pass :(\n";
    }

    test_p_from_rhoh<<<1,1>>>(passed_d);
    hipMemcpy(&passed, passed_d, sizeof(bool), hipMemcpyDeviceToHost);
    if (passed) {
        cout << "p_from_rhoh passed!\n";
    } else {
        cout << "p_from_rhoh did not pass :(\n";
    }

    test_p_from_rho_eps<<<1,1>>>(passed_d);
    hipMemcpy(&passed, passed_d, sizeof(bool), hipMemcpyDeviceToHost);
    if (passed) {
        cout << "p_from_rho_eps passed!\n";
    } else {
        cout << "p_from_rho_eps did not pass :(\n";
    }

    test_hdot<<<1,1>>>(passed_d);
    hipMemcpy(&passed, passed_d, sizeof(bool), hipMemcpyDeviceToHost);
    if (passed) {
        cout << "h_dot passed!\n";
    } else {
        cout << "h_dot did not pass :(\n";
    }


    test_calc_As<<<1,1>>>(passed_d);
    hipMemcpy(&passed, passed_d, sizeof(bool), hipMemcpyDeviceToHost);
    if (passed) {
        cout << "calc_As passed!\n";
    } else {
        cout << "calc_As did not pass :(\n";
    }

    int ntests = 100;
    bool *passed_vec_d, *passed_vec;
    passed_vec = (bool *)malloc(ntests*sizeof(bool));
    hipMalloc((void**)&passed_vec_d, ntests*sizeof(bool));

    float * q_prim, *q_prim_d;
    q_prim = (float *)malloc(5*ntests*sizeof(float));
    hipMalloc((void**)&q_prim_d, 5*ntests*sizeof(float));

    for (int i = 0; i < ntests; i++) {
        q_prim[i*5+0] = 10*r();
        q_prim[i*5+1] = r()-0.5;
        q_prim[i*5+2] = 1.2*r()-0.6;
        q_prim[i*5+3] = 1.2*r()-0.6;
        q_prim[i*5+4] = 15*r();
    }
    hipMemcpy(q_prim_d, q_prim, ntests*5*sizeof(float), hipMemcpyHostToDevice);
    test_cons_to_prim_comp_d<<<ntests,1>>>(passed_vec_d, q_prim_d);
    hipMemcpy(&passed_vec, passed_vec_d, sizeof(bool), hipMemcpyDeviceToHost);

    passed = true;
    for (int i = 0; i < ntests; i++) {
        if (!(passed_vec[i])) {
            passed = false;
            break;
        }
    }

    if (passed) {
        cout << "cons_to_prim_comp_d passed!\n";
    } else {
        cout << "cons_to_prim_comp_d did not pass :(\n";
    }

    free(passed_vec);
    hipFree(passed_vec_d);
    hipFree(q_prim_d);
    free(q_prim);

    hipFree(passed_d);
}
