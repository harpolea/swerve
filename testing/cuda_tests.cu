#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <helper_functions.h>
#include "../Mesh_cuda.h"
#include "../mesh_cuda_kernel.h"
#include "unit_tests.h"

using namespace std;

/*
This does some unit tests on some of the functions in swerve.
*/

bool test_cons_to_prim_comp_d_wrapper() {
    int ntests = 100;
    bool *passed_vec_d, *passed_vec;
    passed_vec = (bool *)malloc(ntests*sizeof(bool));
    hipMalloc((void**)&passed_vec_d, ntests*sizeof(bool));

    float * q_prim, *q_prim_d;
    q_prim = (float *)malloc(6*ntests*sizeof(float));
    hipMalloc((void**)&q_prim_d, 6*ntests*sizeof(float));

    for (int i = 0; i < ntests; i++) {
        q_prim[i*6+0] = 10*r();
        q_prim[i*6+1] = 0.8*r()-0.4;
        q_prim[i*6+2] = r()-0.5;
        q_prim[i*6+3] = r()-0.5;
        q_prim[i*6+4] = 15*r();
        q_prim[i*6+5] = r();
    }
    hipMemcpy(q_prim_d, q_prim, ntests*6*sizeof(float), hipMemcpyHostToDevice);
    test_cons_to_prim_comp_d<<<1,ntests>>>(passed_vec_d, q_prim_d);
    hipMemcpy(passed_vec, passed_vec_d, ntests*sizeof(bool), hipMemcpyDeviceToHost);

    bool passed = true;
    for (int i = 0; i < ntests; i++) {
        if (!(passed_vec[i])) {
            passed = false;
        }
    }

    free(passed_vec);
    hipFree(passed_vec_d);
    hipFree(q_prim_d);
    free(q_prim);

    return passed;
}

bool test_swe_from_compressible() {
    /*
    TODO: FIX THIS TEST
    */
    bool passed = true;
    float gamma_up[] = {0.80999862,  0.0 ,  0.0,  0.0,  0.80999862,
        0.0,  0.0,  0.0,  0.80999862};
    float gamma = 5.0/3.0;
    int kx_offset = 0;
    int ky_offset = 0;
    int nxs[] = {10, 5};
    int nys[] = {1,1};
    int nzs[] = {1, 2};
    const int coarse_level = 0;

    float * gamma_up_d;
    hipMalloc((void**)&gamma_up_d, 9*sizeof(float));
    hipMemcpy(gamma_up_d, gamma_up, 9*sizeof(float), hipMemcpyHostToDevice);

    int matching_indices[] = {1, 10, 0, 0};

    float q[] = {1.0,0.0,0.0,0.0,1.0, 0.0,
                 0.001, 0.0, 0.0, 0.0, 0.001, 0.0,
                 1000., 0.0, 0.0, 0.0, 1000., 0.0,
                 1.05245657, 0.59075458, 0.59075458, 0.59075458, 1.23464966, 0.0,
                 1.03406473, 0.2142144, 0.2142144, 0.0, 0.03634062, 0.0, 1.0,0.0,0.0,0.0,1.0, 0.0,
                  0.001, 0.0, 0.0, 0.0, 0.001, 0.0,
                  1000., 0.0, 0.0, 0.0, 1000., 0.0,
                  1.05245657, 0.59075458, 0.59075458, 0.59075458, 1.23464966, 0.0,
                  1.03406473, 0.2142144, 0.2142144, 0.0, 0.03634062, 0.0};
    float qc[] = {0.39233170120469052, 0.0, -0.0, 0.0,
                     -2.3707704103881642, -0.0, 0.0, 0.0,
                     3.1554338127975452, 0.0, -0.0, 0.0,
                     0.41291207788984285, 0.086914406123075383, 0.086914406123075383, 0.0,
                     0.00068880264357815526, 0.000142453303890117, 0.000142453303890117, 0.0};
    float q_swe[] = {0.39233170120469052, 0.0, -0.0, 0.0,
                     -2.3707704103881642, -0.0, 0.0, 0.0,
                     3.1554338127975452, 0.0, -0.0, 0.0,
                     0.41291207788984285, 0.086914406123075383, 0.086914406123075383, 0.0,
                     0.00068880264357815526, 0.000142453303890117, 0.000142453303890117, 0.0};
    float rho[] = {1.0, 1.0e-3, 1.0e3, 1.0, 1.0,1.0, 1.0e-3, 1.0e3, 1.0, 1.0};

    float * q_d, *q_swe_new, * q_swe_d, *rho_d, *qc_d;
    hipMalloc((void**)&q_d, 6*nxs[1]*nzs[1]*sizeof(float));
    q_swe_new = (float *)malloc(4*nxs[1]*nzs[1]*sizeof(float));
    hipMalloc((void**)&q_swe_d, 4*nxs[1]*nzs[1]*sizeof(float));

    hipMalloc((void**)&rho_d, nxs[1]*nzs[1]*sizeof(float));
    hipMemcpy(q_d, q, 6*nxs[1]*nzs[1]*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(rho_d, rho, nxs[1]*nzs[1]*sizeof(float), hipMemcpyHostToDevice);
    hipMalloc((void**)&qc_d, 4*nxs[0]*nys[0]*nzs[0]*sizeof(float));
    hipMemcpy(qc_d, qc, 4*nxs[0]*nys[0]*nzs[0]*sizeof(float), hipMemcpyHostToDevice);

    int *nxs_d, * nys_d, * nzs_d;
    hipMalloc((void**)&nxs_d, 2*sizeof(int));
    hipMemcpy(nxs_d, nxs, 2*sizeof(int), hipMemcpyHostToDevice);
    hipMalloc((void**)&nys_d, 2*sizeof(int));
    hipMemcpy(nys_d, nys, 2*sizeof(int), hipMemcpyHostToDevice);
    hipMalloc((void**)&nzs_d, 2*sizeof(int));
    hipMemcpy(nzs_d, nzs, 2*sizeof(int), hipMemcpyHostToDevice);

    int * matching_indices_d;
    hipMalloc((void**)&matching_indices_d, 4*sizeof(int));
    hipMemcpy(matching_indices_d, matching_indices, 4*sizeof(int), hipMemcpyHostToDevice);

    dim3 block(nxs[1], nys[1], nzs[1]);

    swe_from_compressible<<<1,block>>>(q_d, q_swe_d,
                                   nxs_d, nys_d, nzs_d, gamma_up_d,
                                   rho_d, gamma, kx_offset, ky_offset,
                                   qc_d, matching_indices_d, coarse_level);

    hipMemcpy(q_swe_new, q_swe_d, 4*nxs[1]*nzs[1]*sizeof(float), hipMemcpyDeviceToHost);

    const float tol = 1.0e-5;
    for (int i = 0; i < nxs[1]; i++) {
        for (int n = 0; n < 4; n++) {
            if ((abs((q_swe[i*4+n] - q_swe_new[i*4+n]) / q_swe[i*4+n]) > tol) && (abs(q_swe[i*4+n] - q_swe_new[i*4+n]) > 0.1*tol)) {
                printf("component %d: %f, %f\n", n, q_swe[i*4+n], q_swe_new[i*4+n]);
                passed = false;
            }
        }
    }

    hipFree(gamma_up_d);
    hipFree(q_d);
    free(q_swe_new);
    hipFree(q_swe_d);
    hipFree(rho_d);

    hipFree(nxs_d);
    hipFree(nys_d);
    hipFree(nzs_d);
    hipFree(matching_indices_d);

    return passed;
}

void run_cuda_tests() {

    bool passed = true;
    bool *passed_d;
    hipMalloc((void**)&passed_d, sizeof(bool));

    test_find_height<<<1,1>>>(passed_d);
    hipMemcpy(&passed, passed_d, sizeof(bool), hipMemcpyDeviceToHost);
    if (passed) {
        cout << "find_height passed!\n";
    } else {
        cout << "find_height did not pass :(\n";
    }

    test_find_pot<<<1,1>>>(passed_d);
    hipMemcpy(&passed, passed_d, sizeof(bool), hipMemcpyDeviceToHost);
    if (passed) {
        cout << "find_pot passed!\n";
    } else {
        cout << "find_pot did not pass :(\n";
    }

    test_rhoh_from_p<<<1,1>>>(passed_d);
    hipMemcpy(&passed, passed_d, sizeof(bool), hipMemcpyDeviceToHost);
    if (passed) {
        cout << "rhoh_from_p passed!\n";
    } else {
        cout << "rhoh_from_p did not pass :(\n";
    }

    test_p_from_rhoh<<<1,1>>>(passed_d);
    hipMemcpy(&passed, passed_d, sizeof(bool), hipMemcpyDeviceToHost);
    if (passed) {
        cout << "p_from_rhoh passed!\n";
    } else {
        cout << "p_from_rhoh did not pass :(\n";
    }

    test_p_from_rho_eps<<<1,1>>>(passed_d);
    hipMemcpy(&passed, passed_d, sizeof(bool), hipMemcpyDeviceToHost);
    if (passed) {
        cout << "p_from_rho_eps passed!\n";
    } else {
        cout << "p_from_rho_eps did not pass :(\n";
    }

    test_hdot<<<1,1>>>(passed_d);
    hipMemcpy(&passed, passed_d, sizeof(bool), hipMemcpyDeviceToHost);
    if (passed) {
        cout << "h_dot passed!\n";
    } else {
        cout << "h_dot did not pass :(\n";
    }

    test_calc_As<<<1,1>>>(passed_d);
    hipMemcpy(&passed, passed_d, sizeof(bool), hipMemcpyDeviceToHost);
    if (passed) {
        cout << "calc_As passed!\n";
    } else {
        cout << "calc_As did not pass :(\n";
    }

    passed = test_cons_to_prim_comp_d_wrapper();
    if (passed) {
        cout << "cons_to_prim_comp_d passed!\n";
    } else {
        cout << "cons_to_prim_comp_d did not pass :(\n";
    }

    test_shallow_water_fluxes<<<1,1>>>(passed_d);
    hipMemcpy(&passed, passed_d, sizeof(bool), hipMemcpyDeviceToHost);
    if (passed) {
        cout << "shallow_water_fluxes passed!\n";
    } else {
        cout << "shallow_water_fluxes did not pass :(\n";
    }

    test_compressible_fluxes<<<1,1>>>(passed_d);
    hipMemcpy(&passed, passed_d, sizeof(bool), hipMemcpyDeviceToHost);
    if (passed) {
        cout << "compressible_fluxes passed!\n";
    } else {
        cout << "compressible_fluxes did not pass :(\n";
    }

    test_p_from_swe<<<1,1>>>(passed_d);
    hipMemcpy(&passed, passed_d, sizeof(bool), hipMemcpyDeviceToHost);
    if (passed) {
        cout << "p_from_swe passed!\n";
    } else {
        cout << "p_from_swe did not pass :(\n";
    }

    passed = test_swe_from_compressible();
    if (passed) {
        cout << "swe_from_compressible passed!\n";
    } else {
        cout << "swe_from_compressible did not pass :(\n";
    }

    hipFree(passed_d);
}
