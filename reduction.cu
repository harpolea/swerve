#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
    Parallel reduction kernels
*/

#ifndef _REDUCE_KERNEL_H_
#define _REDUCE_KERNEL_H_

#include <stdio.h>

// Utility class used to avoid linker errors with extern
// unsized shared memory arrays with templated type
template<class T>
struct SharedMemory
{
    __device__ inline operator       T *()
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }

    __device__ inline operator const T *() const
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }
};

// specialize for double to avoid unaligned memory
// access compile errors
template<>
struct SharedMemory<double>
{
    __device__ inline operator       double *()
    {
        extern __shared__ double __smem_d[];
        return (double *)__smem_d;
    }

    __device__ inline operator const double *() const
    {
        extern __shared__ double __smem_d[];
        return (double *)__smem_d;
    }
};


/*
    This version adds multiple elements per thread sequentially.  This reduces the overall
    cost of the algorithm while keeping the work complexity O(n) and the step complexity O(log n).
    (Brent's Theorem optimization)

    Note, this kernel needs a minimum of 64*sizeof(T) bytes of shared memory.
    In other words if blockSize <= 32, allocate 64*sizeof(T) bytes.
    If blockSize > 32, allocate blockSize*sizeof(T) bytes.
*/
template <class T, unsigned int blockSize, bool nIsPow2>
__global__ void
reduce6(T *g_idata, T *g_odata, unsigned int n)
{
    T *sdata = SharedMemory<T>();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
    unsigned int gridSize = blockSize*2*gridDim.x;

    T mySum = 0;

    // we reduce multiple elements per thread.  The number is determined by the
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    {
        mySum += g_idata[i];

        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (nIsPow2 || i + blockSize < n)
            mySum += g_idata[i+blockSize];

        i += gridSize;
    }

    // each thread puts its local sum into shared memory
    sdata[tid] = mySum;
    __syncthreads();


    // do reduction in shared mem
    if ((blockSize >= 512) && (tid < 256))
    {
        sdata[tid] = mySum = mySum + sdata[tid + 256];
    }

    __syncthreads();

    if ((blockSize >= 256) &&(tid < 128))
    {
            sdata[tid] = mySum = mySum + sdata[tid + 128];
    }

     __syncthreads();

    if ((blockSize >= 128) && (tid <  64))
    {
       sdata[tid] = mySum = mySum + sdata[tid +  64];
    }

    __syncthreads();

#if (__CUDA_ARCH__ >= 300 )
    if ( tid < 32 )
    {
        // Fetch final intermediate sum from 2nd warp
        if (blockSize >=  64) mySum += sdata[tid + 32];
        // Reduce final warp using shuffle
        for (int offset = warpSize/2; offset > 0; offset /= 2)
        {
            mySum += __shfl_down(mySum, offset);
        }
    }
#else
    // fully unroll reduction within a single warp
    if ((blockSize >=  64) && (tid < 32))
    {
        sdata[tid] = mySum = mySum + sdata[tid + 32];
    }

    __syncthreads();

    if ((blockSize >=  32) && (tid < 16))
    {
        sdata[tid] = mySum = mySum + sdata[tid + 16];
    }

    __syncthreads();

    if ((blockSize >=  16) && (tid <  8))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  8];
    }

    __syncthreads();

    if ((blockSize >=   8) && (tid <  4))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  4];
    }

    __syncthreads();

    if ((blockSize >=   4) && (tid <  2))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  2];
    }

    __syncthreads();

    if ((blockSize >=   2) && ( tid <  1))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  1];
    }

    __syncthreads();
#endif

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = mySum;
}


extern "C"
bool isPow2(unsigned int x);


////////////////////////////////////////////////////////////////////////////////
// Wrapper function for kernel launch
////////////////////////////////////////////////////////////////////////////////
template <class T>
void
reduce(int size, int threads, int blocks, T *d_idata, T *d_odata)
{
    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);

    // when there is only one warp per block, we need to allocate two warps
    // worth of shared memory so that we don't index shared memory out of bounds
    int smemSize = (threads <= 32) ? 2 * threads * sizeof(T) : threads * sizeof(T);


    default:
        if (isPow2(size))
        {
            switch (threads)
            {
                case 512:
                    reduce6<T, 512, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                    break;

                case 256:
                    reduce6<T, 256, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                    break;

                case 128:
                    reduce6<T, 128, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                    break;

                case 64:
                    reduce6<T,  64, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                    break;

                case 32:
                    reduce6<T,  32, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                    break;

                case 16:
                    reduce6<T,  16, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                    break;

                case  8:
                    reduce6<T,   8, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                    break;

                case  4:
                    reduce6<T,   4, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                    break;

                case  2:
                    reduce6<T,   2, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                    break;

                case  1:
                    reduce6<T,   1, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                    break;
            }
        }
        else
        {
            switch (threads)
            {
                case 512:
                    reduce6<T, 512, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                    break;

                case 256:
                    reduce6<T, 256, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                    break;

                case 128:
                    reduce6<T, 128, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                    break;

                case 64:
                    reduce6<T,  64, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                    break;

                case 32:
                    reduce6<T,  32, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                    break;

                case 16:
                    reduce6<T,  16, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                    break;

                case  8:
                    reduce6<T,   8, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                    break;

                case  4:
                    reduce6<T,   4, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                    break;

                case  2:
                    reduce6<T,   2, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                    break;

                case  1:
                    reduce6<T,   1, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                    break;
            }
        }

        break;

}

// Instantiate the reduction function for 3 types
template void
reduce<int>(int size, int threads, int blocks,
            int whichKernel, int *d_idata, int *d_odata);

template void
reduce<float>(int size, int threads, int blocks,
              int whichKernel, float *d_idata, float *d_odata);

template void
reduce<double>(int size, int threads, int blocks,
               int whichKernel, double *d_idata, double *d_odata);


#endif // #ifndef _REDUCE_KERNEL_H_
