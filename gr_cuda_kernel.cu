#include "hip/hip_runtime.h"
#ifndef _GR_CUDA_KERNEL_H_
#define _GR_CUDA_KERNEL_H_

#include <stdio.h>
#include "H5Cpp.h"

using namespace std;

// prototypes

dim3 getNumKernels(int nx, int ny, int nlayers, int ng, int *maxBlocks, int *maxThreads);

void getNumBlocksAndThreads(int nx, int ny, int nlayers, int ng, int maxBlocks, int maxThreads, dim3 kernels, dim3 *blocks, dim3 *threads);

unsigned int nextPow2(unsigned int x);

void bcs_fv(float * grid, int nx, int ny, int nlayers, int ng);

void homogeneuous_fv(dim3 kernels, dim3 * threads, dim3 * blocks, float * beta_d, float * gamma_up_d,
       float * Un_d, float * F_d,
       float * qx_p_d, float * qx_m_d, float * qy_p_d, float * qy_m_d,
       float * fx_p_d, float * fx_m_d, float * fy_p_d, float * fy_m_d,
       int nx, int ny, int nlayers, float alpha,
       float dx, float dy, float dt);

void rk3_fv(dim3 kernels, dim3 * threads, dim3 * blocks,
      float * beta_d, float * gamma_up_d, float * Un_d,
      float * F_d, float * Up_d,
      float * qx_p_d, float * qx_m_d, float * qy_p_d, float * qy_m_d,
      float * fx_p_d, float * fx_m_d, float * fy_p_d, float * fy_m_d,
      int nx, int ny, int nlayers, int ng, float alpha,
      float dx, float dy, float dt,
      float * Up_h, float * F_h, float * Un_h);

unsigned int nextPow2(unsigned int x)
{
    --x;
    x |= x >> 1;
    x |= x >> 2;
    x |= x >> 4;
    x |= x >> 8;
    x |= x >> 16;
    return ++x;
}

dim3 getNumKernels(int nx, int ny, int nlayers, int ng, int *maxBlocks, int *maxThreads) {
    /*
    Return the number of kernels needed to run the problem given its size and the constraints of the GPU.
    */
    // won't actually use maxThreads - fix to account for the fact we want something square
    *maxThreads = nlayers * int(sqrt(float(*maxThreads)/nlayers)) * int(sqrt(*maxThreads/nlayers));
    *maxBlocks = int(sqrt(float(*maxBlocks))) * int(sqrt(float(*maxBlocks)));

    //int numBlocks = 0;
    //int numThreads = 0;

    dim3 kernels;

    // calculate number of kernels needed

    if (nx*ny*nlayers > *maxBlocks * *maxThreads) {
        kernels.x = int(ceil(float(nx-2*ng) / (sqrt(float(*maxThreads * *maxBlocks)/nlayers) - 2.0*ng)));
        kernels.y = int(ceil(float(ny-2*ng) / (sqrt(float(*maxThreads * *maxBlocks)/nlayers) - 2.0*ng)));

    } else {

        kernels.x = 1;
        kernels.y = 1;
    }

    return kernels;
}

void getNumBlocksAndThreads(int nx, int ny, int nlayers, int ng, int maxBlocks, int maxThreads, dim3 kernels, dim3 *blocks, dim3 *threads)
{
    /*
    Returns the number of blocks and threads required for each kernel given the size of the problem and the constraints of the device.
    */

    //get device capability, to avoid block/grid size exceed the upper bound
    hipDeviceProp_t prop;
    int device;
    hipGetDevice(&device);
    hipGetDeviceProperties(&prop, device);

    int total = nx*ny*nlayers;

    if ((kernels.x > 1) || (kernels.y > 1)) {
        // initialise
        threads[0].x = 0;
        threads[0].y = 0;
        blocks[0].x = 0;
        blocks[0].y = 0;


        for (int j = 0; j < (kernels.y-1); j++) {
            for (int i = 0; i < (kernels.x-1); i++) {
                threads[j*kernels.x + i].x = int(sqrt(float(maxThreads)/nlayers));
                threads[j*kernels.x + i].y = int(sqrt(float(maxThreads)/nlayers));
                threads[j*kernels.x + i].z = nlayers;

                blocks[j*kernels.x + i].x = int(sqrt(float(maxBlocks)));
                blocks[j*kernels.x + i].y = int(sqrt(float(maxBlocks)));
                blocks[j*kernels.x + i].z = 1;
            }


        }
        // kernels.x-1
        int nx_remaining = nx - threads[0].x * blocks[0].x * (kernels.x - 1);
        for (int j = 0; j < (kernels.y-1); j++) {


            threads[j*kernels.x + kernels.x-1].y =
                int(sqrt(float(maxThreads)/nlayers));
            threads[j*kernels.x + kernels.x-1].z = nlayers;

            threads[j*kernels.x + kernels.x-1].x =
                (nx_remaining < threads[j*kernels.x + kernels.x-1].y) ? nx_remaining : threads[j*kernels.x + kernels.x-1].y;

            blocks[j*kernels.x + kernels.x-1].x = int(ceil(float(nx_remaining) /
                float(threads[j*kernels.x + kernels.x-1].x)));
            blocks[j*kernels.x + kernels.x-1].y = int(sqrt(float(maxBlocks)));
            blocks[j*kernels.x + kernels.x-1].z = 1;
        }

        // kernels.y-1
        int ny_remaining = ny - threads[0].y * blocks[0].y * (kernels.y - 1);
        for (int i = 0; i < (kernels.x-1); i++) {

            threads[(kernels.y-1)*kernels.x + i].x =
                int(sqrt(float(maxThreads)/nlayers));
            threads[(kernels.y-1)*kernels.x + i].y =
                (ny_remaining < threads[(kernels.y-1)*kernels.x + i].x) ? ny_remaining : threads[(kernels.y-1)*kernels.x + i].x;
            threads[(kernels.y-1)*kernels.x + i].z = nlayers;

            blocks[(kernels.y-1)*kernels.x + i].x = int(sqrt(float(maxBlocks)));
            blocks[(kernels.y-1)*kernels.x + i].y = int(ceil(float(ny_remaining) /
                float(threads[(kernels.y-1)*kernels.x + i].y)));
            blocks[(kernels.y-1)*kernels.x + i].z = 1;
        }

        // (kernels.x-1, kernels.y-1)
        threads[(kernels.y-1)*kernels.x + kernels.x-1].x =
            (nx_remaining < int(sqrt(float(maxThreads)/nlayers))) ? nx_remaining : int(sqrt(float(maxThreads/nlayers)));
        threads[(kernels.y-1)*kernels.x + kernels.x-1].y =
            (ny_remaining < int(sqrt(float(maxThreads)/nlayers))) ? ny_remaining : int(sqrt(float(maxThreads)/nlayers));
        threads[(kernels.y-1)*kernels.x + kernels.x-1].z = nlayers;

        blocks[(kernels.y-1)*kernels.x + kernels.x-1].x =
            int(ceil(float(nx_remaining) /
            float(threads[(kernels.y-1)*kernels.x + kernels.x-1].x)));
        blocks[(kernels.y-1)*kernels.x + kernels.x-1].y =
            int(ceil(float(ny_remaining) /
            float(threads[(kernels.y-1)*kernels.x + kernels.x-1].y)));
        blocks[(kernels.y-1)*kernels.x + kernels.x-1].z = 1;

    } else {

        int total_threads = (total < maxThreads*2) ? nextPow2((total + 1)/ 2) : maxThreads;
        threads[0].x = int(floor(sqrt(float(total_threads)/float(nlayers))));
        threads[0].y = int(floor(sqrt(float(total_threads)/float(nlayers))));
        threads[0].z = nlayers;
        total_threads = threads[0].x * threads[0].y * threads[0].z;
        int total_blocks = int(ceil(float(total) / float(total_threads)));

        //printf("total blocks: %i\n", total_blocks);

        blocks[0].x = int(ceil(sqrt(float(total_blocks)/float(nx*ny))*nx));
        blocks[0].y = int(ceil(sqrt(float(total_blocks)/float(nx*ny))*ny));
        blocks[0].z = 1;

        total_blocks = blocks[0].x * blocks[0].y;

        //printf("total blocks: %i\n", total_blocks);

        if ((float)total_threads*total_blocks > (float)prop.maxGridSize[0] * prop.maxThreadsPerBlock)
        {
            printf("n is too large, please choose a smaller number!\n");
        }

        if (total_blocks > prop.maxGridSize[0])
        {
            printf("Grid size <%d> exceeds the device capability <%d>, set block size as %d (original %d)\n",
                   total_blocks, prop.maxGridSize[0], total_threads*2, total_threads);

            blocks[0].x /= 2;
            blocks[0].y /= 2;
            threads[0].x *= 2;
            threads[0].y *= 2;
        }

    }
}



__device__ void bcs(float * grid, int nx, int ny, int nlayers, int kx_offset, int ky_offset) {
    /*
    Enforce boundary conditions on section of grid.
    */
    // outflow
    int x = kx_offset + blockIdx.x * blockDim.x + threadIdx.x;
    int y = ky_offset + blockIdx.y * blockDim.y + threadIdx.y;
    int l = threadIdx.z;

    if ((l < nlayers) && (y < ny) && (x < nx) ) {
        for (int i = 0; i < 4; i++) {
            if (x == 0) {
                grid[((y * nx) * nlayers + l)*4+i] = grid[((y * nx + 1) * nlayers + l)*4+i];
            } else if (x == (nx-1)) {
                grid[((y * nx + (nx-1)) * nlayers + l)*4+i] = grid[((y * nx + (nx-2)) * nlayers + l)*4+i];
            } else if (y == 0) {
                grid[(x * nlayers + l)*4+i] = grid[((nx + x) * nlayers + l)*4+i];
            } else if (y == (ny-1)) {
                grid[(((ny-1) * nx + x) * nlayers + l)*4+i] = grid[(((ny-2) * nx + x) * nlayers + l)*4+i];
            }
        }
    }

}

__device__ void bcs_fv(float * grid, int nx, int ny, int nlayers, int ng, int kx_offset, int ky_offset) {
    /*
    Enforce boundary conditions on section of grid.
    */
    // outflow
    int x = kx_offset + blockIdx.x * blockDim.x + threadIdx.x;
    int y = ky_offset + blockIdx.y * blockDim.y + threadIdx.y;
    int l = threadIdx.z;

    if ((l < nlayers) && (y < ny) && (x < nx) ) {
        for (int i = 0; i < 4; i++) {
            if (x < ng) {
                grid[((y * nx + x) * nlayers + l)*4+i] = grid[((y * nx + ng) * nlayers + l)*4+i];
            } else if (x > (nx-ng-1)) {
                grid[((y * nx + x) * nlayers + l)*4+i] = grid[((y * nx + (nx-ng-1)) * nlayers + l)*4+i];
            } else if (y < ng) {
                grid[((y * nx + x) * nlayers + l)*4+i] = grid[(((ng * nx + x) *  + x) * nlayers + l)*4+i];
            } else if (y > (ny-ng-1)) {
                grid[((y * nx + x) * nlayers + l)*4+i] = grid[(((ny-ng-1) * nx + x) * nlayers + l)*4+i];
            }
        }
    }

}

void bcs_fv(float * grid, int nx, int ny, int nlayers, int ng) {
    /*
    Enforce boundary conditions on section of grid.
    */
    // outflow

    for (int l = 0; l < nlayers; l++) {
        for (int y = 0; y < ny; y++){
            for (int i = 0; i < 4; i++) {
                for (int g = 0; g < ng; g++) {
                    grid[((y * nx + g) * nlayers + l)*4+i] = grid[((y * nx + ng) * nlayers + l)*4+i];

                    grid[((y * nx + (nx-1-g)) * nlayers + l)*4+i] = grid[((y * nx + (nx-1-ng)) * nlayers + l)*4+i];
                }
            }
        }
        for (int x = 0; x < nx; x++){
            for (int i = 0; i < 4; i++) {
                for (int g = 0; g < ng; g++) {
                    grid[((g * nx + x) * nlayers + l)*4+i] = grid[((ng * nx + x) * nlayers + l)*4+i];

                    grid[(((ny-1-g) * nx + x) * nlayers + l)*4+i] = grid[(((ny-1-ng) * nx + x) * nlayers + l)*4+i];
                }
            }

        }
    }
}

__device__ void Jx(float * u, float * beta_d, float * gamma_up_d, float * jx, float alpha) {
    /*
    Calculate Jacobian in the x-direction.
    */

    float W = sqrt((u[1]*u[1] * gamma_up_d[0] +
                2.0 * u[1]* u[2] * gamma_up_d[1] +
                u[2]*u[2] * gamma_up_d[3]) / (u[0]*u[0]) + 1.0);
    //cout << "W = " << W << '\n';
    //cout << "u = " << u[0] << ' ' << u[1] << ' ' << u[2] << '\n';

    float ph = u[0] / W;
    float vx = u[1] / (u[0] * W); // u_down
    float vy = u[2] / (u[0] * W); // v_down

    float qx = vx * gamma_up_d[0] + vy * gamma_up_d[1] - beta_d[0]/alpha;

    float chi = 1.0 / (1.0 - vx*vx * W*W - vy*vy * W*W);

    jx[0*3+0] = qx/chi - vx;
    jx[0*3+1] = (1.0 + vy*vy*W*W)/W;
    jx[0*3+2] = -W * vx * vy;

    jx[1*3+0] = -2.0*pow(W,3)*vx*qx*(vx*vx + vy*vy) + ph*(1.0/W - W*vx*vx);
    jx[1*3+1] = qx * (1.0+W*W*vx*vx + W*W*vy*vy) + 0.5*ph*vx*(vy*vy*W*W-1.0);
    jx[1*3+2] = -vy*ph*(1.0 + 0.5*W*W*vx*vx);

    jx[2*3+0] = -W*vy*(2.0*W*W*qx*(vx*vx+vy*vy) + 0.5*ph*vx);
    jx[2*3+1] = 0.5*ph*vy*(1.0+vy*vy*W*W);
    jx[2*3+2] = qx*(1.0+W*W*vx*vx+W*W*vy*vy) - 0.5*ph*W*W*vx*vy*vy;

    for (int i = 0; i < 9; i++) {
        jx[i] *= chi;
    }
}

__device__ void Jy(float * u, float * beta_d, float * gamma_up_d, float * jy, float alpha) {
    /*
    Calculate Jacobian in the y-direction.
    */

    float W = sqrt((u[1]*u[1] * gamma_up_d[0] +
                2.0 * u[1]* u[2] * gamma_up_d[1] +
                u[2]*u[2] * gamma_up_d[3]) / (u[0]*u[0]) + 1.0);

    float ph = u[0] / W;
    float vx = u[1] / (u[0] * W); // u_down
    float vy = u[2] / (u[0] * W); // v_down

    float qy = vy * gamma_up_d[3] + vx * gamma_up_d[1] - beta_d[1]/alpha;

    float chi = 1.0 / (1.0 - vx*vx * W*W - vy*vy * W*W);

    jy[0] = qy/chi - vx;
    jy[1] = -W * vx * vy;
    jy[2] = (1.0 + vx*vx*W*W)/W;

    jy[1*3] = -W*vx*(2.0*W*W*qy*(vx*vx+vy*vy) + 0.5*ph*vy);
    jy[1*3+1] = qy*(1.0+W*W*vx*vx+W*W*vy*vy) - 0.5*ph*W*W*vx*vx*vy;
    jy[1*3+2] = 0.5*ph*vx*(1.0+vx*vx*W*W);

    jy[2*3+0] = -2.0*pow(W,3)*vy*qy*(vx*vx + vy*vy) + ph*(1.0/W - W*vy*vy);
    jy[2*3+1] = -vx*ph*(1.0 + 0.5*W*W*vy*vy);
    jy[2*3+2] = qy * (1.0+W*W*vx*vx + W*W*vy*vy) + 0.5*ph*vy*(vx*vx*W*W-1.0);

    for (int i = 0; i < 9; i++) {
        jy[i] *= chi;
    }

}

__device__ void calc_Q(float * U, float * rho_d, float * Q_d,
                       int nx, int ny, int nlayers,
                       int kx_offset, int ky_offset) {
    /*
    Calculate heating rate using equation 64 of Spitkovsky 2002.
    */
    int x = kx_offset + blockIdx.x * blockDim.x + threadIdx.x;
    int y = ky_offset + blockIdx.y * blockDim.y + threadIdx.y;
    int l = threadIdx.z;

    // set some constants
    //float kappa = 0.03; // opacity, constant
    //float column_depth = 5.4; // y
    float Y = 1.0; // for simplicity as they do just have eps_3alpha = 0 so that helium abundance remains constant.

    // in this model the scale height represents the temperature

    if ((x > 0) && (x < (nx-1)) && (y > 0) && (y < (ny-1)) && (l < nlayers)) {
        // changed to e^-35 to try and help GPU
        Q_d[(y * nx + x) * nlayers + l] = 3.0e13 * rho_d[l]*rho_d[l] * pow(Y, 3) * exp(-35.0/U[((y * nx + x) * nlayers + l)*4]) / pow(U[((y * nx + x) * nlayers + l)*4], 3); //- 0.4622811 * pow(U[((y * nx + x) * nlayers + l)*4], 4) / (3.0 * kappa * column_depth * column_depth);
    }


}

__global__ void evolve(float * beta_d, float * gamma_up_d,
                     float * Un_d, float * Up, float * U_half,
                     float * sum_phs, float * rho_d, float * Q_d,
                     float mu,
                     int nx, int ny, int nlayers, float alpha,
                     float dx, float dy, float dt,
                     int kx_offset, int ky_offset) {
    /*
    First part of evolution through one timestep.
    */
    int x = kx_offset + blockIdx.x * blockDim.x + threadIdx.x;
    int y = ky_offset + blockIdx.y * blockDim.y + threadIdx.y;
    int l = threadIdx.z;

    //if (x*y*l == 0) {
    //    printf("evolving\n");
    //}

    float *u, *A, *B, *A2, *B2, *AB;

    u = (float *) malloc(3*sizeof(float));
    A = (float *) malloc(9*sizeof(float));
    B = (float *) malloc(9*sizeof(float));
    A2 = (float *) malloc(9*sizeof(float));
    B2 = (float *) malloc(9*sizeof(float));
    AB = (float *) malloc(9*sizeof(float));

    //if (x*y*l == 0) {
        //printf("evolving\n");
    //}

    float d, e, f, g, h;
    float * beta;
    beta = (float *) malloc(2*sizeof(float));

    if ((x > 0) && (x < (nx-1)) && (y > 0) && (y < (ny-1)) && (l < nlayers)) {

        for (int i = 0; i < 4; i++) {
            u[i] = Un_d[((y * nx + x) * nlayers + l)*4+i];
        }
        beta[0] = beta_d[(y * nx + x) * 2];
        beta[1] = beta_d[(y * nx + x) * 2 + 1];

        Jx(u, beta, gamma_up_d, A, alpha);
        Jy(u, beta, gamma_up_d, B, alpha);

        // matrix multiplication
        for (int i = 0; i < 3; i++) {
            for (int j = 0; j < 3; j++) {
                A2[i*3+j] = 0;
                B2[i*3+j] = 0;
                AB[i*3+j] = 0;
                for (int k = 0; k < 3; k++) {
                    A2[i*3+j] += A[i*3+k] * A[k*3+j];
                    B2[i*3+j] += B[i*3+k] * B[k*3+j];
                    AB[i*3+j] += A[i*3+k] * B[k*3+j];
                }
            }
        }

        // going to do matrix calculations to calculate different terms
        for (int i = 0; i < 3; i ++) {
            d = 0;
            e = 0;
            f = 0;
            g = 0;
            h = 0;
            for (int j = 0; j < 4; j++) {
                d += A[i*3+j] *
                    (Un_d[((y * nx + x+1) * nlayers + l)*4+j] -
                    Un_d[((y * nx + x-1) * nlayers + l)*4+j]);

                e += B[i*3+j] *
                    (Un_d[(((y+1) * nx + x) * nlayers + l)*4+j] -
                    Un_d[(((y-1) * nx + x) * nlayers + l)*4+j]);

                f += A2[i*3+j] *
                    (Un_d[((y * nx + x+1) * nlayers + l)*4+j] - 2.0 *
                    Un_d[((y * nx + x) * nlayers + l)*4+j] +
                    Un_d[((y * nx + x-1) * nlayers + l)*4+j]);

                g += B2[i*3+j] *
                    (Un_d[(((y+1) * nx + x) * nlayers + l)*4+j] - 2.0 *
                    Un_d[((y * nx + x) * nlayers + l)*4+j] +
                    Un_d[(((y-1) * nx + x) * nlayers + l)*4+j]);

                h += AB[i*3+j] *
                    (Un_d[(((y+1) * nx + x+1) * nlayers + l)*4+j] -
                    Un_d[(((y-1) * nx + x+1) * nlayers + l)*4+j] -
                    Un_d[(((y+1) * nx + x-1) * nlayers + l)*4+j] +
                    Un_d[(((y-1) * nx + x-1) * nlayers + l)*4+j]);
            }

            Up[((y * nx + x) * nlayers + l) * 4 + i] = u[i] + alpha * (
                    -0.5 * dt/dx * d -
                    0.5 * dt/dy * e +
                    0.5 * dt*dt/(dx*dx) * f +
                    0.5 * dt*dt/(dy*dy) * g -
                    0.25 * dt*dt/(dx*dy) * h);

        }

        //if (isnan(Up[((y * nx + x) * nlayers + l)*4])) {
            //printf("Up is %f! ", Up[((y * nx + x) * nlayers + l)*4]);
        //}


    }

    free(u);
    free(A);
    free(B);
    free(A2);
    free(B2);
    free(AB);
    free(beta);

    __syncthreads();

    // enforce boundary conditions
    bcs(Up, nx, ny, nlayers, kx_offset, ky_offset);

    // copy to U_half
    if ((x < nx) && (y < ny) && (l < nlayers)) {
        for (int i = 0; i < 4; i++) {
            U_half[((y * nx + x) * nlayers + l)*4+i] =
                Up[((y * nx + x) * nlayers + l)*4+i];
        }
    }

    float W = 1.0;

    // do source terms
    if ((x < nx) && (y < ny) && (l < nlayers)) {

        //ph[l] = U_half[((y * nx + x) * nlayers + l)*4];
        //Sx[l] = U_half[((y * nx + x) * nlayers + l)*4+1];
        //Sy[l] = U_half[((y * nx + x) * nlayers + l)*4+2];
        W = sqrt(float((U_half[((y * nx + x) * nlayers + l)*4+1] *
            U_half[((y * nx + x) * nlayers + l)*4+1] * gamma_up_d[0] +
            2.0 * U_half[((y * nx + x) * nlayers + l)*4+1] *
            U_half[((y * nx + x) * nlayers + l)*4+2] *
            gamma_up_d[1] +
            U_half[((y * nx + x) * nlayers + l)*4+2] *
            U_half[((y * nx + x) * nlayers + l)*4+2] *
            gamma_up_d[3]) /
            (U_half[((y * nx + x) * nlayers + l)*4] *
            U_half[((y * nx + x) * nlayers + l)*4]) + 1.0));

        //if (isnan(U_half[((y * nx + x) * nlayers + l)*4])) {
            //printf("ph is %f! ", U_half[((y * nx + x) * nlayers + l)*4]);
        //}
        U_half[((y * nx + x) * nlayers + l)*4] /= W;

    }

    __syncthreads();

    if ((x < nx) && (y < ny) && (l < nlayers)) {

        sum_phs[(y * nx + x) * nlayers + l] = 0.0;


        float sum_qs = 0.0;
        float deltaQx = 0.0;
        float deltaQy = 0.0;

        if (l < (nlayers - 1)) {
            sum_qs += (Q_d[(y * nx + x) * nlayers + l+1] - Q_d[(y * nx + x) * nlayers + l]);
            deltaQx = (Q_d[(y * nx + x) * nlayers + l] + mu) *
                (U_half[((y * nx + x) * nlayers + l)*4+1] -
                 U_half[((y * nx + x) * nlayers + (l+1))*4+1]) /
                 (W*U_half[((y * nx + x) * nlayers + l)*4]);
            deltaQy = (Q_d[(y * nx + x) * nlayers + l] + mu) *
                (U_half[((y * nx + x) * nlayers + l)*4+2] -
                 U_half[((y * nx + x) * nlayers + (l+1))*4+2]) /
                 (W*U_half[((y * nx + x) * nlayers + l)*4]);
        }
        if (l > 0) {
            sum_qs += -rho_d[l-1] / rho_d[l] * (Q_d[(y * nx + x) * nlayers + l] - Q_d[(y * nx + x) * nlayers + l-1]);
            deltaQx = rho_d[l-1] / rho_d[l] *
                (Q_d[(y * nx + x) * nlayers + l] + mu) *
                (U_half[((y * nx + x) * nlayers + l)*4+1] -
                 U_half[((y * nx + x) * nlayers + l-1)*4+1]) /
                 (W*U_half[((y * nx + x) * nlayers + l)*4]);
            deltaQy = rho_d[l-1] / rho_d[l] *
                (Q_d[(y * nx + x) * nlayers + l] + mu) *
                (U_half[((y * nx + x) * nlayers + l)*4+2] -
                 U_half[((y * nx + x) * nlayers + l-1)*4+2]) /
                 (W*U_half[((y * nx + x) * nlayers + l)*4]);
        }

        for (int j = 0; j < l; j++) {
            sum_phs[(y * nx + x) * nlayers + l] += rho_d[j] / rho_d[l] *
                U_half[((y * nx + x) * nlayers + j)*4];
        }
        for (int j = l+1; j < nlayers; j++) {
            sum_phs[(y * nx + x) * nlayers + l] = sum_phs[(y * nx + x) * nlayers + l] +
                U_half[((y * nx + x) * nlayers + j)*4];
        }

        // D
        Up[((y * nx + x) * nlayers + l)*4] += dt * alpha * sum_qs;

        // Sx
        Up[((y * nx + x) * nlayers + l)*4+1] += dt * alpha * (-deltaQx);

        // Sy
        Up[((y * nx + x) * nlayers + l)*4+2] += dt * alpha * (-deltaQy);

    }

}

__global__ void evolve_fv(float * beta_d, float * gamma_up_d,
                     float * Un_d,
                     float * qx_plus_half, float * qx_minus_half,
                     float * qy_plus_half, float * qy_minus_half,
                     float * fx_plus_half, float * fx_minus_half,
                     float * fy_plus_half, float * fy_minus_half,
                     int nx, int ny, int nlayers, float alpha,
                     float dx, float dy, float dt,
                     int kx_offset, int ky_offset) {
    /*
    First part of evolution through one timestep using finite volume methods.
    Reconstructs state vector to cell boundaries using slope limiter
    and calculates fluxes there.

    NOTE: we assume that beta is smooth so can get value at cell boundaries with simple averaging
    */
    int x = kx_offset + blockIdx.x * blockDim.x + threadIdx.x;
    int y = ky_offset + blockIdx.y * blockDim.y + threadIdx.y;
    int l = threadIdx.z;

    int offset = ((y * nx + x) * nlayers + l) * 4;

    if ((x > 0) && (x < (nx-1)) && (y > 0) && (y < (ny-1)) && (l < nlayers)) {

        // x-direction
        for (int i = 0; i < 4; i++) {
            float S_upwind = (Un_d[((y * nx + x+1) * nlayers + l) * 4 + i] -
                Un_d[((y * nx + x) * nlayers + l) * 4 + i]) / dx;
            float S_downwind = (Un_d[((y * nx + x) * nlayers + l) * 4 + i] -
                Un_d[((y * nx + x-1) * nlayers + l) * 4 + i]) / dx;
            float S = 0.5 * (S_upwind + S_downwind); // S_av

            float r = 1.0e5;

            // make sure don't divide by zero
            if (abs(S_downwind) > 1.0e-5) {
                r = S_upwind / S_downwind;
            }

            // MC
            //float phi = max(float(0.0), min(float((2.0 * r) / (1.0 + r)), float(2.0 / (1.0 + r))));
            // superbee
            float phi = 0.0;
            if (r >= 1.0) {
                phi = min(float(2.0), min(r, float(2.0 / (1.0 + r))));
            } else if (r >= 0.5) {
                phi = 1.0;
            } else if (r > 0.0) {
                phi = 2.0 * r;
            }

            S *= phi;

            qx_plus_half[offset + i] = Un_d[offset + i] + S * 0.5 * dx;
            qx_minus_half[offset + i] = Un_d[offset + i] - S * 0.5 * dx;

            // initialise
            fx_plus_half[offset + i] = 0.0;
            fx_minus_half[offset + i] = 0.0;
        }

        // plus half stuff

        float W = sqrt(
            float(qx_plus_half[offset + 1] * qx_plus_half[offset + 1] *
            gamma_up_d[0] +
            2.0 * qx_plus_half[offset + 1] * qx_plus_half[offset + 2] *
            gamma_up_d[1] +
            qx_plus_half[offset + 2] * qx_plus_half[offset + 2] *
            gamma_up_d[3]) /
            (qx_plus_half[offset] * qx_plus_half[offset]) + 1.0);

        float u = qx_plus_half[offset + 1] / (qx_plus_half[offset] * W);
        float v = qx_plus_half[offset + 2] / (qx_plus_half[offset] * W);
        // beta[0] at i+1/2, j
        float beta = 0.5 * (beta_d[(y * nx + x) * 2] + beta_d[(y * nx + x+1) * 2]);
        float qx = u * gamma_up_d[0] + v * gamma_up_d[1] - beta / alpha;

        fx_plus_half[offset] = qx_plus_half[offset] * qx;

        fx_plus_half[offset + 1] = qx_plus_half[offset + 1] * qx +
            0.5 * qx_plus_half[offset] * qx_plus_half[offset] / (W*W);

        fx_plus_half[offset + 2] = qx_plus_half[offset + 2] * qx;

        fx_plus_half[offset + 3] = qx_plus_half[offset + 3] * qx;

        // minus half stuff
        W = sqrt(
            float(qx_minus_half[offset + 1] * qx_minus_half[offset + 1] *
            gamma_up_d[0] +
            2.0 * qx_minus_half[offset + 1] * qx_minus_half[offset + 2] *
            gamma_up_d[1] +
            qx_minus_half[offset + 2] * qx_minus_half[offset + 2] *
            gamma_up_d[3]) /
            (qx_minus_half[offset] * qx_minus_half[offset]) + 1.0);

        u = qx_minus_half[offset + 1] / (qx_minus_half[offset] * W);
        v = qx_minus_half[offset + 2] / (qx_minus_half[offset] * W);
        // beta[0] at i-1/2, j
        beta = 0.5 * (beta_d[(y * nx + x-1) * 2] + beta_d[(y * nx + x) * 2]);
        qx = u * gamma_up_d[0] + v * gamma_up_d[1] - beta / alpha;

        fx_minus_half[offset] = qx_minus_half[offset] * qx;
        fx_minus_half[offset + 1] = qx_minus_half[offset + 1] * qx +
            0.5 * qx_minus_half[offset] * qx_minus_half[offset] / (W*W);
        fx_minus_half[offset + 2] = qx_minus_half[offset + 2] * qx;
        fx_minus_half[offset + 3] = qx_minus_half[offset + 3] * qx;

        // y-direction
        for (int i = 0; i < 4; i++) {
            float S_upwind = (Un_d[(((y+1) * nx + x) * nlayers + l) * 4 + i] -
                Un_d[((y * nx + x) * nlayers + l) * 4 + i]) / dy;
            float S_downwind = (Un_d[((y * nx + x) * nlayers + l) * 4 + i] -
                Un_d[(((y-1) * nx + x) * nlayers + l) * 4 + i]) / dy;
            float S = 0.5 * (S_upwind + S_downwind); // S_av

            float r = 1.0e5;

            // make sure don't divide by zero
            if (abs(S_downwind) > 1.0e-5) {
                r = S_upwind / S_downwind;
            }

            // MC
            //float phi = max(float(0.0), min(float((2.0 * r) / (1.0 + r)), float(2.0 / (1.0 + r))));
            // superbee
            float phi = 0.0;
            if (r >= 1.0) {
                phi = min(float(2.0), min(r, float(2.0 / (1.0 + r))));
            } else if (r >= 0.5) {
                phi = 1.0;
            } else if (r > 0.0) {
                phi = 2.0 * r;
            }

            S *= phi;

            qy_plus_half[offset + i] = Un_d[offset + i] + S * 0.5 * dy;
            qy_minus_half[offset + i] = Un_d[offset + i] - S * 0.5 * dy;

            // initialise
            fy_plus_half[offset + i] = 0.0;
            fy_minus_half[offset + i] = 0.0;
        }

        // plus half stuff

        W = sqrt(
            float(qy_plus_half[offset + 1] * qy_plus_half[offset + 1] *
            gamma_up_d[0] +
            2.0 * qy_plus_half[offset + 1] * qy_plus_half[offset + 2] *
            gamma_up_d[1] +
            qy_plus_half[offset + 2] * qy_plus_half[offset + 2] *
            gamma_up_d[3]) /
            (qy_plus_half[offset] * qy_plus_half[offset]) + 1.0);

        u = qy_plus_half[offset + 1] / (qy_plus_half[offset] * W);
        v = qy_plus_half[offset + 2] / (qy_plus_half[offset] * W);
        // beta[1] at i, j+1/2
        beta = 0.5 * (beta_d[((y+1) * nx + x) * 2 + 1] + beta_d[(y * nx + x) * 2 + 1]);
        float qy = v * gamma_up_d[3] + u * gamma_up_d[1] - beta / alpha;

        fy_plus_half[offset] = qy_plus_half[offset] * qy;
        fy_plus_half[offset + 1] = qy_plus_half[offset + 1] * qy;
        fy_plus_half[offset + 2] = qy_plus_half[offset + 2] * qy +
            0.5 * qy_plus_half[offset] * qy_plus_half[offset] / (W*W);
        fy_plus_half[offset + 3] = qy_plus_half[offset + 3] * qy;

        // minus half stuff
        W = sqrt(
            float(qy_minus_half[offset+1] * qy_minus_half[offset+1] *
            gamma_up_d[0] +
            2.0 * qy_minus_half[offset + 1] * qy_minus_half[offset + 2] *
            gamma_up_d[1] +
            qy_minus_half[offset+2] * qy_minus_half[offset + 2] *
            gamma_up_d[3]) /
            (qy_minus_half[offset]*qy_minus_half[offset]) + 1.0);

        u = qy_minus_half[offset + 1] / (qy_minus_half[offset] * W);
        v = qy_minus_half[offset + 2] / (qy_minus_half[offset] * W);
        // beta[1] at i, j-1/2
        beta = 0.5 * (beta_d[((y-1) * nx + x) * 2 + 1] + beta_d[(y * nx + x) * 2 + 1]);
        qy = v * gamma_up_d[3] + u * gamma_up_d[1] - beta / alpha;

        fy_minus_half[offset] = qy_minus_half[offset] * qy;
        fy_minus_half[offset + 1] = qy_minus_half[offset + 1] * qy;
        fy_minus_half[offset + 2] = qy_minus_half[offset + 2] * qy +
            0.5 * qy_minus_half[offset] * qy_minus_half[offset] / (W*W);
        fy_minus_half[offset + 3] = qy_minus_half[offset + 3] * qy;
    }

}

__global__ void evolve_fv_fluxes(float * F,
                     float * qx_plus_half, float * qx_minus_half,
                     float * qy_plus_half, float * qy_minus_half,
                     float * fx_plus_half, float * fx_minus_half,
                     float * fy_plus_half, float * fy_minus_half,
                     int nx, int ny, int nlayers, float alpha,
                     float dx, float dy, float dt,
                     int kx_offset, int ky_offset) {
    /*
    Calculates fluxes in finite volume evolution by solving the Riemann
    problem at the cell boundaries.
    */
    int x = kx_offset + blockIdx.x * blockDim.x + threadIdx.x;
    int y = ky_offset + blockIdx.y * blockDim.y + threadIdx.y;
    int l = threadIdx.z;

    // do fluxes
    if ((x > 0) && (x < (nx-1)) && (y > 0) && (y < (ny-1)) && (l < nlayers)) {
        for (int i = 0; i < 4; i++) {
            float fx_m = 0.5 * (
                fx_plus_half[((y * nx + x-1) * nlayers + l) * 4 + i] +
                fx_minus_half[((y * nx + x) * nlayers + l) * 4 + i] +
                qx_plus_half[((y * nx + x-1) * nlayers + l) * 4 + i] -
                qx_minus_half[((y * nx + x) * nlayers + l) * 4 + i]);

            float fx_p = 0.5 * (
                fx_plus_half[((y * nx + x) * nlayers + l) * 4 + i] +
                fx_minus_half[((y * nx + x+1) * nlayers + l) * 4 + i] +
                qx_plus_half[((y * nx + x) * nlayers + l) * 4 + i] -
                qx_minus_half[((y * nx + x+1) * nlayers + l) * 4 + i]);

            float fy_m = 0.5 * (
                fy_plus_half[(((y-1) * nx + x) * nlayers + l) * 4 + i] +
                fy_minus_half[((y * nx + x) * nlayers + l) * 4 + i] +
                qy_plus_half[(((y-1) * nx + x) * nlayers + l) * 4 + i] -
                qy_minus_half[((y * nx + x) * nlayers + l) * 4 + i]);


            float fy_p = 0.5 * (
                fy_plus_half[((y * nx + x) * nlayers + l) * 4 + i] +
                fy_minus_half[(((y+1) * nx + x) * nlayers + l) * 4 + i] +
                qy_plus_half[((y * nx + x) * nlayers + l) * 4 + i] -
                qy_minus_half[(((y+1) * nx + x) * nlayers + l) * 4 + i]);

            F[((y * nx + x) * nlayers + l)*4 + i] =
                -((1.0/dx) * alpha * (fx_p - fx_m) +
                (1.0/dy) * alpha * (fy_p - fy_m));

        }

    }
}

__global__ void evolve_fv_heating(float * gamma_up_d,
                     float * Un_d, float * Up, float * U_half,
                     float * qx_plus_half, float * qx_minus_half,
                     float * qy_plus_half, float * qy_minus_half,
                     float * fx_plus_half, float * fx_minus_half,
                     float * fy_plus_half, float * fy_minus_half,
                     float * sum_phs, float * rho_d, float * Q_d,
                     float mu,
                     int nx, int ny, int nlayers, float alpha,
                     float dx, float dy, float dt,
                     int kx_offset, int ky_offset) {
    /*
    Does the heating part of the evolution.
    */
    int x = kx_offset + blockIdx.x * blockDim.x + threadIdx.x;
    int y = ky_offset + blockIdx.y * blockDim.y + threadIdx.y;
    int l = threadIdx.z;


    // copy to U_half
    if ((x < nx) && (y < ny) && (l < nlayers)) {
        for (int i = 0; i < 4; i++) {
            U_half[((y * nx + x) * nlayers + l)*4+i] =
                Up[((y * nx + x) * nlayers + l)*4+i];
        }
    }

    // calculate Q
    calc_Q(Up, rho_d, Q_d, nx, ny, nlayers, kx_offset, ky_offset);

    float W = 1.0;


    // do source terms
    if ((x < nx) && (y < ny) && (l < nlayers)) {

        //ph[l] = U_half[((y * nx + x) * nlayers + l)*4];
        //Sx[l] = U_half[((y * nx + x) * nlayers + l)*4+1];
        //Sy[l] = U_half[((y * nx + x) * nlayers + l)*4+2];
        W = sqrt(float((U_half[((y * nx + x) * nlayers + l)*4+1] *
            U_half[((y * nx + x) * nlayers + l)*4+1] * gamma_up_d[0] +
            2.0 * U_half[((y * nx + x) * nlayers + l)*4+1] *
            U_half[((y * nx + x) * nlayers + l)*4+2] *
            gamma_up_d[1] +
            U_half[((y * nx + x) * nlayers + l)*4+2] *
            U_half[((y * nx + x) * nlayers + l)*4+2] *
            gamma_up_d[3]) /
            (U_half[((y * nx + x) * nlayers + l)*4] *
            U_half[((y * nx + x) * nlayers + l)*4]) + 1.0));

        //if (isnan(U_half[((y * nx + x) * nlayers + l)*4])) {
            //printf("ph is %f! ", U_half[((y * nx + x) * nlayers + l)*4]);
        //}
        U_half[((y * nx + x) * nlayers + l)*4] /= W;

    }

    __syncthreads();

    if ((x < nx) && (y < ny) && (l < nlayers)) {

        sum_phs[(y * nx + x) * nlayers + l] = 0.0;


        float sum_qs = 0.0;
        float deltaQx = 0.0;
        float deltaQy = 0.0;

        if (l < (nlayers - 1)) {
            sum_qs += (Q_d[(y * nx + x) * nlayers + l+1] - Q_d[(y * nx + x) * nlayers + l]);
            deltaQx = (Q_d[(y * nx + x) * nlayers + l] + mu) *
                (U_half[((y * nx + x) * nlayers + l)*4+1] -
                 U_half[((y * nx + x) * nlayers + (l+1))*4+1]) /
                 (W*U_half[((y * nx + x) * nlayers + l)*4]);
            deltaQy = (Q_d[(y * nx + x) * nlayers + l] + mu) *
                (U_half[((y * nx + x) * nlayers + l)*4+2] -
                 U_half[((y * nx + x) * nlayers + (l+1))*4+2]) /
                 (W*U_half[((y * nx + x) * nlayers + l)*4]);
        }
        if (l > 0) {
            sum_qs += -rho_d[l-1] / rho_d[l] * (Q_d[(y * nx + x) * nlayers + l] - Q_d[(y * nx + x) * nlayers + l-1]);
            deltaQx = rho_d[l-1] / rho_d[l] *
                (Q_d[(y * nx + x) * nlayers + l] + mu) *
                (U_half[((y * nx + x) * nlayers + l)*4+1] -
                 U_half[((y * nx + x) * nlayers + l-1)*4+1]) /
                 (W*U_half[((y * nx + x) * nlayers + l)*4]);
            deltaQy = rho_d[l-1] / rho_d[l] *
                (Q_d[(y * nx + x) * nlayers + l] + mu) *
                (U_half[((y * nx + x) * nlayers + l)*4+2] -
                 U_half[((y * nx + x) * nlayers + l-1)*4+2]) /
                 (W*U_half[((y * nx + x) * nlayers + l)*4]);
        }

        for (int j = 0; j < l; j++) {
            sum_phs[(y * nx + x) * nlayers + l] += rho_d[j] / rho_d[l] *
                U_half[((y * nx + x) * nlayers + j)*4];
        }
        for (int j = l+1; j < nlayers; j++) {
            sum_phs[(y * nx + x) * nlayers + l] = sum_phs[(y * nx + x) * nlayers + l] +
                U_half[((y * nx + x) * nlayers + j)*4];
        }

        // D
        Up[((y * nx + x) * nlayers + l)*4] += dt * alpha * sum_qs;

        // Sx
        Up[((y * nx + x) * nlayers + l)*4+1] += dt * alpha * (-deltaQx);

        // Sy
        Up[((y * nx + x) * nlayers + l)*4+2] += dt * alpha * (-deltaQy);

        // zeta
        Up[((y * nx + x) * nlayers + l)*4+3] += -dt * alpha * Q_d[(y * nx + x) * nlayers + l] * rho_d[l];

    }

}


__global__ void evolve2(float * gamma_up_d,
                     float * Un_d, float * Up, float * U_half,
                     float * sum_phs, float * rho_d, float * Q_d,
                     float mu,
                     int nx, int ny, int nlayers, float alpha,
                     float dx, float dy, float dt,
                     int kx_offset, int ky_offset) {
    /*
    Adds buoyancy terms.
    */
    int x = kx_offset + blockIdx.x * blockDim.x + threadIdx.x;
    int y = ky_offset + blockIdx.y * blockDim.y + threadIdx.y;
    int l = threadIdx.z;

    if ((x > 0) && (x < (nx-1)) && (y > 0) && (y < (ny-1)) && (l < nlayers)) {

        float a = dt * alpha *
            U_half[((y * nx + x) * nlayers + l)*4] * (0.5 / dx) * (sum_phs[(y * nx + x+1) * nlayers + l] -
            sum_phs[(y * nx + x-1) * nlayers + l]);

        if (abs(a) < 0.9 * dx / dt) {
            //printf("a is %f! ", a);
            Up[((y * nx + x) * nlayers + l)*4+1] = Up[((y * nx + x) * nlayers + l)*4+1] - a;
        }

        a = dt * alpha *
            U_half[((y * nx + x) * nlayers + l)*4] * (0.5 / dy) *
            (sum_phs[((y+1) * nx + x) * nlayers + l] -
             sum_phs[((y-1) * nx + x) * nlayers + l]);

        if (abs(a) < 0.9 * dy / dt) {
            //printf("a is %f! ", a);
            Up[((y * nx + x) * nlayers + l)*4+2] = Up[((y * nx + x) * nlayers + l)*4+2] - a;
        }


    }

    __syncthreads();

    bcs(Up, nx, ny, nlayers, kx_offset, ky_offset);

    // copy back to grid
    if ((x < nx) && (y < ny) && (l < nlayers)) {
        for (int i = 0; i < 4; i++) {
            Un_d[((y * nx + x) * nlayers + l)*4+i] =
                Up[((y * nx + x) * nlayers + l)*4+i];
        }
    }


}

void homogeneuous_fv(dim3 kernels, dim3 * threads, dim3 * blocks, float * beta_d, float * gamma_up_d,
       float * Un_d, float * F_d,
       float * qx_p_d, float * qx_m_d, float * qy_p_d, float * qy_m_d,
       float * fx_p_d, float * fx_m_d, float * fy_p_d, float * fy_m_d,
       int nx, int ny, int nlayers, float alpha,
       float dx, float dy, float dt) {
    /*
    Solves the homogeneous part of the equation (ie the bit without source terms).
    */

    int kx_offset = 0;
    int ky_offset = 0;

    for (int j = 0; j < kernels.y; j++) {
       kx_offset = 0;
       for (int i = 0; i < kernels.x; i++) {
           evolve_fv<<<blocks[j * kernels.x + i], threads[j * kernels.x + i]>>>(beta_d, gamma_up_d, Un_d,
                  qx_p_d, qx_m_d, qy_p_d, qy_m_d,
                  fx_p_d, fx_m_d, fy_p_d, fy_m_d,
                  nx, ny, nlayers, alpha,
                  dx, dy, dt, kx_offset, ky_offset);
          kx_offset += blocks[j * kernels.x + i].x * threads[j * kernels.x + i].x;
       }
       ky_offset += blocks[j * kernels.x].y * threads[j * kernels.x].y;
    }

    ky_offset = 0;

    for (int j = 0; j < kernels.y; j++) {
       kx_offset = 0;
       for (int i = 0; i < kernels.x; i++) {
           evolve_fv_fluxes<<<blocks[j * kernels.x + i],
                              threads[j * kernels.x + i]>>>(
                  F_d,
                  qx_p_d, qx_m_d, qy_p_d, qy_m_d,
                  fx_p_d, fx_m_d, fy_p_d, fy_m_d,
                  nx, ny, nlayers, alpha,
                  dx, dy, dt, kx_offset, ky_offset);

           kx_offset += blocks[j * kernels.x + i].x * threads[j * kernels.x + i].x;
       }
       ky_offset += blocks[j * kernels.x].y * threads[j * kernels.x].y;
    }


}

void rk3_fv(dim3 kernels, dim3 * threads, dim3 * blocks,
       float * beta_d, float * gamma_up_d, float * Un_d,
       float * F_d, float * Up_d,
       float * qx_p_d, float * qx_m_d, float * qy_p_d, float * qy_m_d,
       float * fx_p_d, float * fx_m_d, float * fy_p_d, float * fy_m_d,
       int nx, int ny, int nlayers, int ng, float alpha,
       float dx, float dy, float dt,
       float * Up_h, float * F_h, float * Un_h) {
    /*
    Integrates the homogeneous part of the ODE in time using RK3.
    */

    // u1 = un + dt * F(un)
    homogeneuous_fv(kernels, threads, blocks,
          beta_d, gamma_up_d, Un_d, F_d,
          qx_p_d, qx_m_d, qy_p_d, qy_m_d,
          fx_p_d, fx_m_d, fy_p_d, fy_m_d,
          nx, ny, nlayers, alpha,
          dx, dy, dt);

    // copy back flux
    hipMemcpy(F_h, F_d, nx*ny*nlayers*4*sizeof(float), hipMemcpyDeviceToHost);
    bcs_fv(F_h, nx, ny, nlayers, ng);

    for (int y = 0; y < ny; y++) {
        for (int x = 0; x < nx; x++) {
            for (int l = 0; l < nlayers; l++) {
                for (int i = 0; i < 4; i++) {
                    Up_h[((y * nx + x) * nlayers + l) * 4 + i] = Un_h[((y * nx + x) * nlayers + l) * 4 + i] + dt * F_h[((y * nx + x) * nlayers + l) * 4 + i];
                }
            }
        }
    }

    // enforce boundaries and copy back
    bcs_fv(Up_h, nx, ny, nlayers, ng);
    hipMemcpy(Un_d, Up_h, nx*ny*nlayers*4*sizeof(float), hipMemcpyHostToDevice);

    // u2 = 0.25 * (3*un + u1 + dt*F(u1))
    homogeneuous_fv(kernels, threads, blocks,
          beta_d, gamma_up_d, Un_d, F_d,
          qx_p_d, qx_m_d, qy_p_d, qy_m_d,
          fx_p_d, fx_m_d, fy_p_d, fy_m_d,
          nx, ny, nlayers, alpha,
          dx, dy, dt);

    // copy back flux
    hipMemcpy(F_h, F_d, nx*ny*nlayers*4*sizeof(float), hipMemcpyDeviceToHost);
    bcs_fv(F_h, nx, ny, nlayers, ng);

    for (int y = 0; y < ny; y++) {
        for (int x = 0; x < nx; x++) {
            for (int l = 0; l < nlayers; l++) {
                for (int i = 0; i < 4; i++) {
                    Up_h[((y * nx + x) * nlayers + l) * 4 + i] = 0.25 * (
                        3.0 * Un_h[((y * nx + x) * nlayers + l) * 4 + i] +
                        Up_h[((y * nx + x) * nlayers + l) * 4 + i] +
                        dt * F_h[((y * nx + x) * nlayers + l) * 4 + i]);
                }
            }
        }
    }

    // enforce boundaries and copy back
    bcs_fv(Up_h, nx, ny, nlayers, ng);
    hipMemcpy(Un_d, Up_h, nx*ny*nlayers*4*sizeof(float), hipMemcpyHostToDevice);

    // un+1 = (1/3) * (un + 2*u2 + 2*dt*F(u2))
    homogeneuous_fv(kernels, threads, blocks,
          beta_d, gamma_up_d, Un_d, F_d,
          qx_p_d, qx_m_d, qy_p_d, qy_m_d,
          fx_p_d, fx_m_d, fy_p_d, fy_m_d,
          nx, ny, nlayers, alpha,
          dx, dy, dt);

    // copy back flux
    hipMemcpy(F_h, F_d, nx*ny*nlayers*4*sizeof(float), hipMemcpyDeviceToHost);
    bcs_fv(F_h, nx, ny, nlayers, ng);

    for (int y = 0; y < ny; y++) {
        for (int x = 0; x < nx; x++) {
            for (int l = 0; l < nlayers; l++) {
                for (int i = 0; i < 4; i++) {
                    Up_h[((y * nx + x) * nlayers + l) * 4 + i] = (1/3.0) * (
                        Un_h[((y * nx + x) * nlayers + l) * 4 + i] +
                        2.0*Up_h[((y * nx + x) * nlayers + l) * 4 + i] +
                        2.0*dt * F_h[((y * nx + x) * nlayers + l) * 4 + i]);
                }
            }
        }
    }

    // enforce boundaries
    bcs_fv(Up_h, nx, ny, nlayers, ng);

    hipMemcpy(Up_d, Up_h, nx*ny*nlayers*4*sizeof(float), hipMemcpyHostToDevice);

}


void cuda_run(float * beta, float * gamma_up, float * Un_h,
         float * rho, float * Q, float mu, int nx, int ny, int nlayers, int ng,
         int nt, float alpha, float dx, float dy, float dt, int dprint, char * filename) {
    /*
    Evolve system through nt timesteps, saving data to filename every dprint timesteps.
    */

    bool finite_volume = true;

    // set up GPU stuff
    int count;
    hipGetDeviceCount(&count);

    //int size = 3 * nx * ny * nlayers;
    int maxThreads = 256;
    int maxBlocks = 256; //64;

    dim3 kernels = getNumKernels(nx, ny, nlayers, ng, &maxBlocks, &maxThreads);

    dim3 *blocks = new dim3[kernels.x*kernels.y];
    dim3 *threads = new dim3[kernels.x*kernels.y];

    getNumBlocksAndThreads(nx, ny, nlayers, ng, maxBlocks, maxThreads, kernels, blocks, threads);
    //int numBlocks = blocks.x * blocks.y * blocks.z;

    printf("kernels: (%i, %i)\n", kernels.x, kernels.y);

    for (int i = 0; i < kernels.x*kernels.y; i++) {
        printf("blocks: (%i, %i, %i) , threads: (%i, %i, %i)\n",
               blocks[i].x, blocks[i].y, blocks[i].z,
               threads[i].x, threads[i].y, threads[i].z);
    }

    // copy
    float * beta_d;
    float * gamma_up_d;
    float * Un_d;
    float * rho_d;
    float * Q_d;

    // allocate memory on device
    hipMalloc((void**)&beta_d, 2*nx*ny*sizeof(float));
    hipMalloc((void**)&gamma_up_d, 4*sizeof(float));
    hipMalloc((void**)&Un_d, nx*ny*nlayers*4*sizeof(float));
    hipMalloc((void**)&rho_d, nlayers*sizeof(float));
    hipMalloc((void**)&Q_d, nlayers*nx*ny*sizeof(float));

    // copy stuff to GPU
    hipMemcpy(beta_d, beta, 2*nx*ny*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gamma_up_d, gamma_up, 4*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Un_d, Un_h, nx*ny*nlayers*4*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(rho_d, rho, nlayers*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Q_d, Q, nlayers*nx*ny*sizeof(float), hipMemcpyHostToDevice);

    float *Up_d, *U_half_d, *sum_phs_d;
    hipMalloc((void**)&Up_d, nlayers*nx*ny*4*sizeof(float));
    hipMalloc((void**)&U_half_d, nlayers*nx*ny*4*sizeof(float));
    hipMalloc((void**)&sum_phs_d, nlayers*nx*ny*sizeof(float));

    float *qx_p_d, *qx_m_d, *qy_p_d, *qy_m_d, *fx_p_d, *fx_m_d, *fy_p_d, *fy_m_d;
    float *Up_h = new float[nlayers*nx*ny*4];
    float *F_h = new float[nlayers*nx*ny*4];

    if (finite_volume) {
        hipMalloc((void**)&qx_p_d, nlayers*nx*ny*4*sizeof(float));
        hipMalloc((void**)&qx_m_d, nlayers*nx*ny*4*sizeof(float));
        hipMalloc((void**)&qy_p_d, nlayers*nx*ny*4*sizeof(float));
        hipMalloc((void**)&qy_m_d, nlayers*nx*ny*4*sizeof(float));
        hipMalloc((void**)&fx_p_d, nlayers*nx*ny*4*sizeof(float));
        hipMalloc((void**)&fx_m_d, nlayers*nx*ny*4*sizeof(float));
        hipMalloc((void**)&fy_p_d, nlayers*nx*ny*4*sizeof(float));
        hipMalloc((void**)&fy_m_d, nlayers*nx*ny*4*sizeof(float));
    }

    if (strcmp(filename, "na") != 0) {

        // create file
        hid_t outFile = H5Fcreate(filename, H5F_ACC_TRUNC, H5P_DEFAULT, H5P_DEFAULT);

        // create dataspace
        int ndims = 5;
        hsize_t dims[] = {hsize_t((nt+1)/dprint+1), hsize_t(ny), hsize_t(nx), hsize_t(nlayers), 4};
        hid_t file_space = H5Screate_simple(ndims, dims, NULL);

        hid_t plist = H5Pcreate(H5P_DATASET_CREATE);
        H5Pset_layout(plist, H5D_CHUNKED);
        hsize_t chunk_dims[] = {1, hsize_t(ny), hsize_t(nx), hsize_t(nlayers), 4};
        H5Pset_chunk(plist, ndims, chunk_dims);

        // create dataset
        hid_t dset = H5Dcreate(outFile, "SwerveOutput", H5T_NATIVE_FLOAT, file_space, H5P_DEFAULT, plist, H5P_DEFAULT);

        H5Pclose(plist);

        // make a memory dataspace
        hid_t mem_space = H5Screate_simple(ndims, chunk_dims, NULL);

        // select a hyperslab
        //printf("hyperslab selection\n");
        file_space = H5Dget_space(dset);
        hsize_t start[] = {0, 0, 0, 0, 0};
        hsize_t hcount[] = {1, hsize_t(ny), hsize_t(nx), hsize_t(nlayers), 4};
        H5Sselect_hyperslab(file_space, H5S_SELECT_SET, start, NULL, hcount, NULL);
        //printf("writing\n");
        // write to dataset
        printf("Printing t = %i\n", 0);
        H5Dwrite(dset, H5T_NATIVE_FLOAT, mem_space, file_space, H5P_DEFAULT, Un_h);
        // close file dataspace
        //printf("wrote\n");
        H5Sclose(file_space);

        for (int t = 0; t < nt; t++) {

            int kx_offset = 0;
            int ky_offset = 0;

            if (finite_volume) {

                rk3_fv(kernels, threads, blocks,
                    beta_d, gamma_up_d, Un_d, U_half_d, Up_d,
                    qx_p_d, qx_m_d, qy_p_d, qy_m_d,
                    fx_p_d, fx_m_d, fy_p_d, fy_m_d,
                    nx, ny, nlayers, ng, alpha,
                    dx, dy, dt, Up_h, F_h, Un_h);

                for (int j = 0; j < kernels.y; j++) {
                    kx_offset = 0;
                    for (int i = 0; i < kernels.x; i++) {
                        evolve_fv_heating<<<blocks[j * kernels.x + i], threads[j * kernels.x + i]>>>(
                               gamma_up_d, Un_d,
                               Up_d, U_half_d,
                               qx_p_d, qx_m_d, qy_p_d, qy_m_d,
                               fx_p_d, fx_m_d, fy_p_d, fy_m_d,
                               sum_phs_d, rho_d, Q_d, mu,
                               nx, ny, nlayers, alpha,
                               dx, dy, dt, kx_offset, ky_offset);
                        kx_offset += blocks[j * kernels.x + i].x * threads[j * kernels.x + i].x;
                    }
                    ky_offset += blocks[j * kernels.x].y * threads[j * kernels.x].y;
                }


            } else {
                for (int j = 0; j < kernels.y; j++) {
                    kx_offset = 0;
                    for (int i = 0; i < kernels.x; i++) {
                        evolve<<<blocks[j * kernels.x + i], threads[j * kernels.x + i]>>>(beta_d, gamma_up_d, Un_d,
                               Up_d, U_half_d, sum_phs_d, rho_d, Q_d, mu,
                               nx, ny, nlayers, alpha,
                               dx, dy, dt, kx_offset, ky_offset);
                        kx_offset += blocks[j * kernels.x + i].x * threads[j * kernels.x + i].x;
                    }
                    ky_offset += blocks[j * kernels.x].y * threads[j * kernels.x].y;
                }
            }

            kx_offset = 0;
            ky_offset = 0;

            for (int j = 0; j < kernels.y; j++) {
                kx_offset = 0;
                for (int i = 0; i < kernels.x; i++) {
                    evolve2<<<blocks[j * kernels.x + i], threads[j * kernels.x + i]>>>(gamma_up_d, Un_d,
                           Up_d, U_half_d, sum_phs_d, rho_d, Q_d, mu,
                           nx, ny, nlayers, alpha,
                           dx, dy, dt, kx_offset, ky_offset);
                    kx_offset += blocks[j * kernels.x + i].x * threads[j * kernels.x + i].x;
                }
                ky_offset += blocks[j * kernels.x].y * threads[j * kernels.x].y;
            }

            hipDeviceSynchronize();

            hipError_t err = hipGetLastError();

            if (err != hipSuccess)
                printf("Error: %s\n", hipGetErrorString(err));

            if (finite_volume) {
                // boundaries
                hipMemcpy(Un_h, Un_d, nx*ny*nlayers*4*sizeof(float), hipMemcpyDeviceToHost);
                bcs_fv(Un_h, nx, ny, nlayers, ng);
                hipMemcpy(Un_d, Un_h, nx*ny*nlayers*4*sizeof(float), hipMemcpyHostToDevice);
            }


            if ((t+1) % dprint == 0) {
                printf("Printing t = %i\n", t+1);

                if (finite_volume == false) {
                    // copy stuff back
                    hipMemcpy(Un_h, Un_d, nx*ny*nlayers*4*sizeof(float), hipMemcpyDeviceToHost);
                }

                // select a hyperslab
                file_space = H5Dget_space(dset);
                hsize_t start[] = {hsize_t((t+1)/dprint), 0, 0, 0, 0};
                hsize_t hcount[] = {1, hsize_t(ny), hsize_t(nx), hsize_t(nlayers), 4};
                H5Sselect_hyperslab(file_space, H5S_SELECT_SET, start, NULL, hcount, NULL);
                // write to dataset
                H5Dwrite(dset, H5T_NATIVE_FLOAT, mem_space, file_space, H5P_DEFAULT, Un_h);
                // close file dataspae
                H5Sclose(file_space);
            }
        }
        H5Sclose(mem_space);
        H5Fclose(outFile);

    } else { // don't print
        for (int t = 0; t < nt; t++) {



            //if (t % 50 == 0) {
                //printf("t =  %i\n", t);
            //}
            int kx_offset = 0;
            int ky_offset = 0;

            if (finite_volume) {

                rk3_fv(kernels, threads, blocks,
                    beta_d, gamma_up_d, Un_d, U_half_d, Up_d,
                    qx_p_d, qx_m_d, qy_p_d, qy_m_d,
                    fx_p_d, fx_m_d, fy_p_d, fy_m_d,
                    nx, ny, nlayers, ng, alpha,
                    dx, dy, dt, Up_h, F_h, Un_h);

                for (int j = 0; j < kernels.y; j++) {
                    kx_offset = 0;
                    for (int i = 0; i < kernels.x; i++) {
                        evolve_fv_heating<<<blocks[j * kernels.x + i], threads[j * kernels.x + i]>>>(
                               gamma_up_d, Un_d,
                               Up_d, U_half_d,
                               qx_p_d, qx_m_d, qy_p_d, qy_m_d,
                               fx_p_d, fx_m_d, fy_p_d, fy_m_d,
                               sum_phs_d, rho_d, Q_d, mu,
                               nx, ny, nlayers, alpha,
                               dx, dy, dt, kx_offset, ky_offset);
                        kx_offset += blocks[j * kernels.x + i].x * threads[j * kernels.x + i].x;
                    }
                    ky_offset += blocks[j * kernels.x].y * threads[j * kernels.x].y;
                }


            } else {
                for (int j = 0; j < kernels.y; j++) {
                    kx_offset = 0;
                    for (int i = 0; i < kernels.x; i++) {
                        evolve<<<blocks[j * kernels.x + i], threads[j * kernels.x + i]>>>(beta_d, gamma_up_d, Un_d,
                               Up_d, U_half_d, sum_phs_d, rho_d, Q_d, mu,
                               nx, ny, nlayers, alpha,
                               dx, dy, dt, kx_offset, ky_offset);
                        kx_offset += blocks[j * kernels.x + i].x * threads[j * kernels.x + i].x;
                    }
                    ky_offset += blocks[j * kernels.x].y * threads[j * kernels.x].y;
                }
            }

            kx_offset = 0;
            ky_offset = 0;

            for (int j = 0; j < kernels.y; j++) {
                kx_offset = 0;
                for (int i = 0; i < kernels.x; i++) {
                    evolve2<<<blocks[j * kernels.x + i], threads[j * kernels.x + i]>>>(gamma_up_d, Un_d,
                           Up_d, U_half_d, sum_phs_d, rho_d, Q_d, mu,
                           nx, ny, nlayers, alpha,
                           dx, dy, dt, kx_offset, ky_offset);
                    kx_offset += blocks[j * kernels.x + i].x * threads[j * kernels.x + i].x;
                }
                ky_offset += blocks[j * kernels.x].y * threads[j * kernels.x].y;
            }

            hipDeviceSynchronize();

            hipError_t err = hipGetLastError();

            if (err != hipSuccess)
                printf("Error: %s\n", hipGetErrorString(err));

            if ((t+1) % dprint == 0) {
                printf("Printing t = %i\n", t+1);
                // copy stuff back
                hipMemcpy(Un_h, Un_d, nx*ny*nlayers*4*sizeof(float), hipMemcpyDeviceToHost);

            }
        }
    }


    // delete some stuff
    hipFree(beta_d);
    hipFree(gamma_up_d);
    hipFree(Un_d);
    hipFree(rho_d);
    hipFree(Q_d);
    hipFree(Up_d);
    hipFree(U_half_d);
    hipFree(sum_phs_d);

    if (finite_volume) {
        hipFree(qx_p_d);
        hipFree(qx_m_d);
        hipFree(qy_p_d);
        hipFree(qy_m_d);
        hipFree(fx_p_d);
        hipFree(fx_m_d);
        hipFree(fy_p_d);
        hipFree(fy_m_d);
    }

    //delete[] Un_h;
    delete[] threads;
    delete[] blocks;

    delete[] Up_h;
    delete[] F_h;
}


#endif
