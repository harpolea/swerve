#ifndef _GR_CUDA_KERNEL_H_
#define _GR_CUDA_KERNEL_H_

#include <stdio.h>

void getNumBlocksAndThreads(int nx, int ny, int nlayers, int maxBlocks, int maxThreads, dim3 &blocks, dim3 &threads);

unsigned int nextPow2(unsigned int x);


// TODO: GET RID OF THIS
//void __syncthreads() {}

unsigned int nextPow2(unsigned int x)
{
    --x;
    x |= x >> 1;
    x |= x >> 2;
    x |= x >> 4;
    x |= x >> 8;
    x |= x >> 16;
    return ++x;
}

void getNumBlocksAndThreads(int nx, int ny, int nlayers, int maxBlocks, int maxThreads, dim3 &blocks, dim3 &threads)
{

    //get device capability, to avoid block/grid size exceed the upper bound
    hipDeviceProp_t prop;
    int device;
    hipGetDevice(&device);
    hipGetDeviceProperties(&prop, device);

    int total = nx*ny*nlayers;

    int total_threads = (total < maxThreads*2) ? nextPow2((total + 1)/ 2) : maxThreads;
    threads.x = int(floor(sqrt(float(total_threads)/float(nlayers))));
    threads.y = int(floor(sqrt(float(total_threads)/float(nlayers))));
    threads.z = nlayers;
    total_threads = threads.x * threads.y * threads.z;
    int total_blocks = int(ceil(float(total) / float(total_threads)));

    //printf("total blocks: %i\n", total_blocks);

    blocks.x = int(ceil(sqrt(float(total_blocks)/float(nx*ny))*nx));
    blocks.y = int(ceil(sqrt(float(total_blocks)/float(nx*ny))*ny));

    total_blocks = blocks.x * blocks.y;

    //printf("total blocks: %i\n", total_blocks);

    if ((float)total_threads*total_blocks > (float)prop.maxGridSize[0] * prop.maxThreadsPerBlock)
    {
        printf("n is too large, please choose a smaller number!\n");
    }

    if (total_blocks > prop.maxGridSize[0])
    {
        printf("Grid size <%d> exceeds the device capability <%d>, set block size as %d (original %d)\n",
               total_blocks, prop.maxGridSize[0], total_threads*2, total_threads);

        blocks.x /= 2;
        blocks.y /= 2;
        threads.x *= 2;
        threads.y *= 2;
    }
}



__device__ void bcs(float * grid, int nx, int ny, int nlayers) {
    // outflow

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int l = threadIdx.z;

    if ((l < nlayers) && (y < ny) && (x < nx) ) {
        for (int i = 0; i < 3; i++) {
            if (x == 0) {
                grid[((y * nx) * nlayers + l)*3+i] = grid[((y * nx + 1) * nlayers + l)*3+i];
            } else if (x == (nx-1)) {
                grid[((y * nx + (nx-1)) * nlayers + l)*3+i] = grid[((y * nx + (nx-2)) * nlayers + l)*3+i];
            } else if (y == 0) {
                grid[(x * nlayers + l)*3+i] = grid[((nx + x) * nlayers + l)*3+i];
            } else if (y == (ny-1)) {
                grid[(((ny-1) * nx + x) * nlayers + l)*3+i] = grid[(((ny-2) * nx + x) * nlayers + l)*3+i];
            }
        }
    }

}

__device__ void Jx(float * u, float * beta_d, float * gamma_up_d, float * jx, float alpha) {

    float W = sqrt((u[1]*u[1] * gamma_up_d[0] +
                2.0 * u[1]* u[2] * gamma_up_d[1] +
                u[2]*u[2] * gamma_up_d[3]) / (u[0]*u[0]) + 1.0);
    //cout << "W = " << W << '\n';
    //cout << "u = " << u[0] << ' ' << u[1] << ' ' << u[2] << '\n';

    float ph = u[0] / W;
    float vx = u[1] / (u[0] * W); // u_down
    float vy = u[2] / (u[0] * W); // v_down

    float qx = vx * gamma_up_d[0] + vy * gamma_up_d[1] - beta_d[0]/alpha;

    float chi = 1.0 / (1.0 - vx*vx * W*W - vy*vy * W*W);

    jx[0*3+0] = qx/chi - vx;
    jx[0*3+1] = (1.0 + vy*vy*W*W)/W;
    jx[0*3+2] = -W * vx * vy;

    jx[1*3+0] = -2.0*pow(W,3)*vx*qx*(vx*vx + vy*vy) + ph*(1.0/W - W*vx*vx);
    jx[1*3+1] = qx * (1.0+W*W*vx*vx + W*W*vy*vy) + 0.5*ph*vx*(vy*vy*W*W-1.0);
    jx[1*3+2] = -vy*ph*(1.0 + 0.5*W*W*vx*vx);

    jx[2*3+0] = -W*vy*(2.0*W*W*qx*(vx*vx+vy*vy) + 0.5*ph*vx);
    jx[2*3+1] = 0.5*ph*vy*(1.0+vy*vy*W*W);
    jx[2*3+2] = qx*(1.0+W*W*vx*vx+W*W*vy*vy) - 0.5*ph*W*W*vx*vy*vy;

    for (int i = 0; i < 9; i++) {
        jx[i] *= chi;
    }
}

__device__ void Jy(float * u, float * beta_d, float * gamma_up_d, float * jy, float alpha) {

    float W = sqrt((u[1]*u[1] * gamma_up_d[0] +
                2.0 * u[1]* u[2] * gamma_up_d[1] +
                u[2]*u[2] * gamma_up_d[3]) / (u[0]*u[0]) + 1.0);

    float ph = u[0] / W;
    float vx = u[1] / (u[0] * W); // u_down
    float vy = u[2] / (u[0] * W); // v_down

    float qy = vy * gamma_up_d[3] + vx * gamma_up_d[1] - beta_d[1]/alpha;

    float chi = 1.0 / (1.0 - vx*vx * W*W - vy*vy * W*W);

    jy[0] = qy/chi - vx;
    jy[1] = -W * vx * vy;
    jy[2] = (1.0 + vx*vx*W*W)/W;

    jy[1*3] = -W*vx*(2.0*W*W*qy*(vx*vx+vy*vy) + 0.5*ph*vy);
    jy[1*3+1] = qy*(1.0+W*W*vx*vx+W*W*vy*vy) - 0.5*ph*W*W*vx*vx*vy;
    jy[1*3+2] = 0.5*ph*vx*(1.0+vx*vx*W*W);

    jy[2*3+0] = -2.0*pow(W,3)*vy*qy*(vx*vx + vy*vy) + ph*(1.0/W - W*vy*vy);
    jy[2*3+1] = -vx*ph*(1.0 + 0.5*W*W*vy*vy);
    jy[2*3+2] = qy * (1.0+W*W*vx*vx + W*W*vy*vy) + 0.5*ph*vy*(vx*vx*W*W-1.0);

    for (int i = 0; i < 9; i++) {
        jy[i] *= chi;
    }

}

__global__ void evolve(float * beta_d, float * gamma_up_d,
                     float * Un_d, float * Up, float * U_half,
                     float * sum_phs, float * rho_d, float * Q_d,
                     int nx, int ny, int nlayers, float alpha,
                     float dx, float dy, float dt) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int l = threadIdx.z;

    //if (x*y*l == 0) {
    //    printf("evolving\n");
    //}

    float *u, *A, *B, *A2, *B2, *AB;

    u = (float *) malloc(3*sizeof(float));
    A = (float *) malloc(9*sizeof(float));
    B = (float *) malloc(9*sizeof(float));
    A2 = (float *) malloc(9*sizeof(float));
    B2 = (float *) malloc(9*sizeof(float));
    AB = (float *) malloc(9*sizeof(float));

    //if (x*y*l == 0) {
        //printf("evolving\n");
    //}

    float d, e, f, g, h;

    if ((x > 0) && (x < (nx-1)) && (y > 0) && (y < (ny-1)) && (l < nlayers)) {

        for (int i = 0; i < 3; i++) {
            u[i] = Un_d[((y * nx + x) * nlayers + l)*3+i];
        }

        Jx(u, beta_d, gamma_up_d, A, alpha);
        Jy(u, beta_d, gamma_up_d, B, alpha);

        // matrix multiplication
        for (int i = 0; i < 3; i++) {
            for (int j = 0; j < 3; j++) {
                A2[i*3+j] = 0;
                B2[i*3+j] = 0;
                AB[i*3+j] = 0;
                for (int k = 0; k < 3; k++) {
                    A2[i*3+j] += A[i*3+k] * A[k*3+j];
                    B2[i*3+j] += B[i*3+k] * B[k*3+j];
                    AB[i*3+j] += A[i*3+k] * B[k*3+j];
                }
            }
        }

        // going to do matrix calculations to calculate different terms
        for (int i = 0; i < 3; i ++) {
            d = 0;
            e = 0;
            f = 0;
            g = 0;
            h = 0;
            for (int j = 0; j < 3; j++) {
                d += A[i*3+j] *
                    (Un_d[((y * nx + x+1) * nlayers + l)*3+j] -
                    Un_d[((y * nx + x-1) * nlayers + l)*3+j]);

                e += B[i*3+j] *
                    (Un_d[(((y+1) * nx + x) * nlayers + l)*3+j] -
                    Un_d[(((y-1) * nx + x) * nlayers + l)*3+j]);

                f += A2[i*3+j] *
                    (Un_d[((y * nx + x+1) * nlayers + l)*3+j] - 2.0 *
                    Un_d[((y * nx + x) * nlayers + l)*3+j] +
                    Un_d[((y * nx + x-1) * nlayers + l)*3+j]);

                g += B2[i*3+j] *
                    (Un_d[(((y+1) * nx + x) * nlayers + l)*3+j] - 2.0 *
                    Un_d[((y * nx + x) * nlayers + l)*3+j] +
                    Un_d[(((y-1) * nx + x) * nlayers + l)*3+j]);

                h += AB[i*3+j] *
                    (Un_d[(((y+1) * nx + x+1) * nlayers + l)*3+j] -
                    Un_d[(((y-1) * nx + x+1) * nlayers + l)*3+j] -
                    Un_d[(((y+1) * nx + x-1) * nlayers + l)*3+j] +
                    Un_d[(((y-1) * nx + x-1) * nlayers + l)*3+j]);
            }

            Up[((y * nx + x) * nlayers + l) * 3 + i] = u[i] + alpha * (
                    -0.5 * dt/dx * d -
                    0.5 * dt/dy * e +
                    0.5 * dt*dt/(dx*dx) * f +
                    0.5 * dt*dt/(dy*dy) * g -
                    0.25 * dt*dt/(dx*dy) * h);

        }

        //if (isnan(Up[((y * nx + x) * nlayers + l)*3])) {
            //printf("Up is %f! ", Up[((y * nx + x) * nlayers + l)*3]);
        //}


    }

    free(u);
    free(A);
    free(B);
    free(A2);
    free(B2);
    free(AB);

    __syncthreads();

    // enforce boundary conditions
    bcs(Up, nx, ny, nlayers);

    // copy to U_half
    if ((x < nx) && (y < ny) && (l < nlayers)) {
        for (int i = 0; i < 3; i++) {
            U_half[((y * nx + x) * nlayers + l)*3+i] =
                Up[((y * nx + x) * nlayers + l)*3+i];
        }
    }

    float W = 1.0;

    // do source terms
    if ((x < nx) && (y < ny) && (l < nlayers)) {

        //ph[l] = U_half[((y * nx + x) * nlayers + l)*3];
        //Sx[l] = U_half[((y * nx + x) * nlayers + l)*3+1];
        //Sy[l] = U_half[((y * nx + x) * nlayers + l)*3+2];
        W = sqrt(float((U_half[((y * nx + x) * nlayers + l)*3+1] *
            U_half[((y * nx + x) * nlayers + l)*3+1] * gamma_up_d[0] +
            2.0 * U_half[((y * nx + x) * nlayers + l)*3+1] *
            U_half[((y * nx + x) * nlayers + l)*3+2] *
            gamma_up_d[1] +
            U_half[((y * nx + x) * nlayers + l)*3+2] *
            U_half[((y * nx + x) * nlayers + l)*3+2] *
            gamma_up_d[3]) /
            (U_half[((y * nx + x) * nlayers + l)*3] *
            U_half[((y * nx + x) * nlayers + l)*3]) + 1.0));

        //if (isnan(U_half[((y * nx + x) * nlayers + l)*3])) {
            //printf("ph is %f! ", U_half[((y * nx + x) * nlayers + l)*3]);
        //}
        U_half[((y * nx + x) * nlayers + l)*3] /= W;

    }

    __syncthreads();

    if ((x < nx) && (y < ny) && (l < nlayers)) {

        sum_phs[(y * nx + x) * nlayers + l] = 0.0;

        /*
        float sum_qs = 0.0;
        float deltaQx = 0.0;
        float deltaQy = 0.0;

        if (l < (nlayers - 1)) {
            sum_qs += -rho_d[l+1] / rho_d[l] * abs(Q_d[l+1] - Q_d[l]);
            deltaQx = rho_d[l+1] / rho_d[l] *
                max(float(0.0), Q_d[l] - Q_d[l+1]) *
                (U_half[((y * nx + x) * nlayers + l)*3+1] -
                 U_half[((y * nx + x) * nlayers + (l+1))*3+1]) /
                 U_half[((y * nx + x) * nlayers + l)*3];
            deltaQy = rho_d[l+1] / rho_d[l] *
                max(float(0.0), Q_d[l] - Q_d[l+1]) *
                (U_half[((y * nx + x) * nlayers + l)*3+2] -
                 U_half[((y * nx + x) * nlayers + (l+1))*3+2]) /
                 U_half[((y * nx + x) * nlayers + l)*3];
        }
        if (l > 0) {
            sum_qs += abs(Q_d[l] - Q_d[l-1]);
            deltaQx = max(float(0.0), Q_d[l] - Q_d[l-1]) *
                (U_half[((y * nx + x) * nlayers + l)*3+1] -
                 U_half[((y * nx + x) * nlayers + l-1)*3+1]) /
                 U_half[((y * nx + x) * nlayers + l)*3];
            deltaQy = max(float(0.0), Q_d[l] - Q_d[l-1]) *
                (U_half[((y * nx + x) * nlayers + l)*3+2] -
                 U_half[((y * nx + x) * nlayers + l-1)*3+2]) /
                 U_half[((y * nx + x) * nlayers + l)*3];
        }
        */
        for (int j = 0; j < l; j++) {
            sum_phs[(y * nx + x) * nlayers + l] += rho_d[j] / rho_d[l] *
                U_half[((y * nx + x) * nlayers + j)*3];
            //sum_phs[(y * nx + x) * nlayers + l] = sum_phs[(y * nx + x) * nlayers + l] +
            //    U_half[((y * nx + x) * nlayers + j)*3];
        }
        for (int j = l+1; j < nlayers; j++) {
            sum_phs[(y * nx + x) * nlayers + l] = sum_phs[(y * nx + x) * nlayers + l] +
                U_half[((y * nx + x) * nlayers + j)*3];
        }
        /*
        // D
        Up[((y * nx + x) * nlayers + l)*3] += dt * alpha * sum_qs;

        // Sx
        Up[((y * nx + x) * nlayers + l)*3+1] += dt * alpha *
            U_half[((y * nx + x) * nlayers + l)*3] * (-deltaQx);

        // Sy
        Up[((y * nx + x) * nlayers + l)*3+2] += dt * alpha *
            U_half[((y * nx + x) * nlayers + l)*3] * (-deltaQy);
        */
    }

}

    // code works with this bit commented out.
__global__ void evolve2(float * beta_d, float * gamma_up_d,
                     float * Un_d, float * Up, float * U_half,
                     float * sum_phs, float * rho_d, float * Q_d,
                     int nx, int ny, int nlayers, float alpha,
                     float dx, float dy, float dt) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int l = threadIdx.z;


    if ((x > 0) && (x < (nx-1)) && (y > 0) && (y < (ny-1)) && (l < nlayers)) {

        float a = dt * alpha *
            U_half[((y * nx + x) * nlayers + l)*3] * (0.5 / dx) * (sum_phs[(y * nx + x+1) * nlayers + l] -
            sum_phs[(y * nx + x-1) * nlayers + l]);

        //printf("a: %f ", a);

        if (abs(a) < 0.9 * dx / dt) {
            //printf("a is %f! ", a);
            Up[((y * nx + x) * nlayers + l)*3+1] = Up[((y * nx + x) * nlayers + l)*3+1] - a;
        }

        a = dt * alpha *
            U_half[((y * nx + x) * nlayers + l)*3] * (0.5 / dy) *
            (sum_phs[((y+1) * nx + x) * nlayers + l] -
             sum_phs[((y-1) * nx + x) * nlayers + l]);

        if (abs(a) < 0.9 * dy / dt) {
            //printf("a is %f! ", a);
            Up[((y * nx + x) * nlayers + l)*3+2] = Up[((y * nx + x) * nlayers + l)*3+2] - a;
        }
        /*
        // Sx
        Up[((y * nx + x) * nlayers + l)*3+1] -= dt * alpha *
            U_half[((y * nx + x) * nlayers + l)*3] * (0.5 / dx) *
            (sum_phs[(y * nx + x+1) * nlayers + l] -
             sum_phs[(y * nx + x-1) * nlayers + l]);

        // Sy
        Up[((y * nx + x) * nlayers + l)*3+2] -= dt * alpha *
            U_half[((y * nx + x) * nlayers + l)*3] * (0.5 / dy) *
            (sum_phs[((y+1) * nx + x) * nlayers + l] -
             sum_phs[((y-1) * nx + x) * nlayers + l]);
        */

    }

    __syncthreads();

    bcs(Up, nx, ny, nlayers);

    // copy back to grid
    if ((x < nx) && (y < ny) && (l < nlayers)) {
        for (int i = 0; i < 3; i++) {
            Un_d[((y * nx + x) * nlayers + l)*3+i] =
                Up[((y * nx + x) * nlayers + l)*3+i];
        }
    }


    //if (x*y*l == 0) {
        //printf("finished evolving\n");
    //}

}

void cuda_run(float * beta, float * gamma_up, float * U_grid,
         float * rho, float * Q, int nx, int ny, int nlayers,
         int nt, float alpha, float dx, float dy, float dt) {


    // set up GPU stuff
    int count;
    hipGetDeviceCount(&count);
    //dim3 threadsPerBlock(20,20,nlayers);
    //dim3 numBlocks(nx/threadsPerBlock.x,ny/threadsPerBlock.y,1);

    //int size = 3 * nx * ny * nlayers;
    int maxThreads = 256;
    int maxBlocks = 256; //64;

    //int numBlocks = 0;
    //int numThreads = 0;

    dim3 threads;//PerBlock;(int(floor(sqrt(numThreads/nlayers))), int(floor(sqrt(numThreads/nlayers))), nlayers);
    dim3 blocks;//grid;(numBlocks, 1, 1);

    getNumBlocksAndThreads(nx, ny, nlayers, maxBlocks, maxThreads, blocks, threads);

    //int numBlocks = blocks.x * blocks.y * blocks.z;

    printf("blocks: %i, %i, %i , threads: %i, %i, %i\n", blocks.x, blocks.y, blocks.z, threads.x, threads.y, threads.z);


    // allocate Un memory
    float * Un_h = (float *) malloc(nx*ny*nlayers*3*sizeof(float));

    // copy U_grid stuff
    for (int i = 0; i < nx*ny*nlayers*3; i++) {
        Un_h[i] = U_grid[i];
    }

    // copy

    float * beta_d;
    float * gamma_up_d;
    float * Un_d;
    float * rho_d;
    float * Q_d;

    // allocate memory on device
    hipMalloc((void**)&beta_d, 2*sizeof(float));
    hipMalloc((void**)&gamma_up_d, 4*sizeof(float));
    hipMalloc((void**)&Un_d, nx*ny*nlayers*3*sizeof(float));
    hipMalloc((void**)&rho_d, nlayers*sizeof(float));
    hipMalloc((void**)&Q_d, nlayers*sizeof(float));

    // copy stuff to GPU
    hipMemcpy(beta_d, beta, 2*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gamma_up_d, gamma_up, 4*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Un_d, Un_h, nx*ny*nlayers*3*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(rho_d, rho, nlayers*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Q_d, Q, nlayers*sizeof(float), hipMemcpyHostToDevice);

    float *Up_d, *U_half_d, *sum_phs_d;
    hipMalloc((void**)&Up_d, nlayers*nx*ny*3*sizeof(float));
    hipMalloc((void**)&U_half_d, nlayers*nx*ny*3*sizeof(float));
    hipMalloc((void**)&sum_phs_d, nlayers*nx*ny*sizeof(float));

    for (int t = 0; t < nt; t++) {

        if (t % 50 == 0) {
            printf("t =  %i\n", t);
        }

        evolve<<<blocks, threads>>>(beta_d, gamma_up_d, Un_d,
               Up_d, U_half_d, sum_phs_d, rho_d, Q_d,
               nx, ny, nlayers, alpha,
               dx, dy, dt);

        evolve2<<<blocks, threads>>>(beta_d, gamma_up_d, Un_d,
                Up_d, U_half_d, sum_phs_d, rho_d, Q_d,
                nx, ny, nlayers, alpha,
                dx, dy, dt);

        hipDeviceSynchronize();

        hipError_t err = hipGetLastError();

        if (err != hipSuccess)
            printf("Error: %s\n", hipGetErrorString(err));

        // copy stuff back
        hipMemcpy(Un_h, Un_d, nx*ny*nlayers*3*sizeof(float), hipMemcpyDeviceToHost);

        // save to U_grid
        for (int i = 0; i < nx*ny*nlayers*3; i++) {
            U_grid[(t+1)*nx*ny*nlayers*3 + i] = Un_h[i];
        }
    }


    // delete some stuff
    hipFree(beta_d);
    hipFree(gamma_up_d);
    hipFree(Un_d);
    hipFree(rho_d);
    hipFree(Q_d);
    hipFree(Up_d);
    hipFree(U_half_d);
    hipFree(sum_phs_d);

    free(Un_h);
}


#endif
