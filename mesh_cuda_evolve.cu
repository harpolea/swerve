#include "hip/hip_runtime.h"
/**
File containing routines which model the evolution.
**/

__global__ void evolve_fv(float * Un_d, flux_func_ptr flux_func,
                     float * qx_plus_half, float * qx_minus_half,
                     float * qy_plus_half, float * qy_minus_half,
                     float * fx_plus_half, float * fx_minus_half,
                     float * fy_plus_half, float * fy_minus_half,
                     int nx, int ny, int nz, int vec_dim, float alpha0,
                     float gamma, float zmin, float dz, float R,
                     int kx_offset, int ky_offset) {
    /**
    First part of evolution through one timestep using finite volume methods.
    Reconstructs state vector to cell boundaries using slope limiter
    and calculates fluxes there.

    NOTE: we assume that beta is smooth so can get value at cell boundaries with simple averaging

    Parameters
    ----------
    Un_d : float *
        state vector at each grid point in each layer
    flux_func : flux_func_ptr
        pointer to function to be used to calulate fluxes
    qx_plus_half, qx_minus_half : float *
        state vector reconstructed at right and left boundaries
    qy_plus_half, qy_minus_half : float *
        state vector reconstructed at top and bottom boundaries
    fx_plus_half, fx_minus_half : float *
        flux vector at right and left boundaries
    fy_plus_half, fy_minus_half : float *
        flux vector at top and bottom boundaries
    nx, ny, nz : int
        dimensions of grid
    alpha, gamma : float
        lapse function and adiabatic index
    kx_offset, ky_offset : int
        x, y offset for current kernel
    */

    int x = kx_offset + blockIdx.x * blockDim.x + threadIdx.x;
    int y = ky_offset + blockIdx.y * blockDim.y + threadIdx.y;
    int z = threadIdx.z;

    int offset = ((z * ny + y) * nx + x) * vec_dim;

    if ((x > 0) && (x < (nx-1)) && (y > 0) && (y < (ny-1)) && (z < nz)) {

        float * q_p, *q_m, * f;
        q_p = (float *)malloc(vec_dim * sizeof(float));
        q_m = (float *)malloc(vec_dim * sizeof(float));
        f = (float *)malloc(vec_dim * sizeof(float));

        // x-direction
        for (int i = 0; i < vec_dim; i++) {
            float S_upwind = (Un_d[((z * ny + y) * nx + x+1) * vec_dim + i] -
                Un_d[((z * ny + y) * nx + x) * vec_dim + i]);
            float S_downwind = (Un_d[((z * ny + y) * nx + x) * vec_dim + i] -
                Un_d[((z * ny + y) * nx + x-1) * vec_dim + i]);
            float S = 0.5 * (S_upwind + S_downwind); // S_av

            float r = 1.0e6;

            // make sure don't divide by zero
            if (abs(S_downwind) > 1.0e-7) {
                r = S_upwind / S_downwind;
            }

            S *= phi(r);

            q_p[i] = Un_d[offset + i] + S * 0.5;
            q_m[i] = Un_d[offset + i] - S * 0.5;
        }

        // fluxes
        flux_func(q_p, f, 0, alpha0, gamma, zmin, dz, nz, z, R);

        for (int i = 0; i < vec_dim; i++) {
            qx_plus_half[offset + i] = q_p[i];
            fx_plus_half[offset + i] = f[i];
        }

        flux_func(q_m, f, 0, alpha0, gamma, zmin, dz, nz, z, R);

        for (int i = 0; i < vec_dim; i++) {
            qx_minus_half[offset + i] = q_m[i];
            fx_minus_half[offset + i] = f[i];
            //if (nan_check(q_p[i]) || nan_check(q_m[i]) || nan_check(fx_plus_half[offset + i]) || nan_check(fx_minus_half[offset + i])) printf("(%d, %d, %d) i: %d, qx_p: %f, qx_m: %f, fx_p: %f, fx_m: %f\n", x, y, z, i, q_p[i], q_m[i], fx_plus_half[offset + i], fx_minus_half[offset + i]);
        }

        // y-direction
        for (int i = 0; i < vec_dim; i++) {
            float S_upwind = (Un_d[((z * ny + y+1) * nx + x) * vec_dim + i] -
                Un_d[((z * ny + y) * nx + x) * vec_dim + i]);
            float S_downwind = (Un_d[((z * ny + y) * nx + x) * vec_dim + i] -
                Un_d[((z * ny + y-1) * nx + x) * vec_dim + i]);
            float S = 0.5 * (S_upwind + S_downwind); // S_av

            float r = 1.0e6;

            // make sure don't divide by zero
            if (abs(S_downwind) > 1.0e-7) {
                r = S_upwind / S_downwind;
            }

            S *= phi(r);

            q_p[i] = Un_d[offset + i] + S * 0.5;
            q_m[i] = Un_d[offset + i] - S * 0.5;
        }

        // fluxes

        flux_func(q_p, f, 1, alpha0, gamma, zmin, dz, nz, z, R);

        for (int i = 0; i < vec_dim; i++) {
            qy_plus_half[offset + i] = q_p[i];
            fy_plus_half[offset + i] = f[i];
        }

        flux_func(q_m, f, 1, alpha0, gamma, zmin, dz, nz, z, R);

        for (int i = 0; i < vec_dim; i++) {
            qy_minus_half[offset + i] = q_m[i];
            fy_minus_half[offset + i] = f[i];
            //if (nan_check(q_p[i]) || nan_check(q_m[i])) printf("(%d, %d, %d) i: %d, qy_p: %f, qy_m: %f\n", x, y, z, i, q_p[i], q_m[i]);
        }

        free(q_p);
        free(q_m);
        free(f);
    }
}

__global__ void evolve_z(float * Un_d, flux_func_ptr flux_func,
                     float * qz_plus_half, float * qz_minus_half,
                     float * fz_plus_half, float * fz_minus_half,
                     int nx, int ny, int nz, int vec_dim, float alpha0,
                     float gamma, float zmin, float dz, float R,
                     int kx_offset, int ky_offset) {
    /**
    First part of evolution through one timestep using finite volume methods.
    Reconstructs state vector to cell boundaries using slope limiter
    and calculates fluxes there.

    NOTE: we assume that beta is smooth so can get value at cell boundaries with simple averaging

    Parameters
    ----------
    Un_d : float *
        state vector at each grid point in each layer
    flux_func : flux_func_ptr
        pointer to function to be used to calculate fluxes
    qz_plus_half, qz_minus_half : float *
        state vector reconstructed at top and bottom boundaries
    fz_plus_half, fz_minus_half : float *
        flux vector at top and bottom boundaries
    nx, ny, nz : int
        dimensions of grid
    vec_dim : int
        dimension of state vector
    alpha, gamma : float
        lapse function and adiabatic index
    kx_offset, ky_offset : int
        x, y offset for current kernel
    */

    int x = kx_offset + blockIdx.x * blockDim.x + threadIdx.x;
    int y = ky_offset + blockIdx.y * blockDim.y + threadIdx.y;
    int z = threadIdx.z;
    int offset = ((z * ny + y) * nx + x) * vec_dim;

    // NOTE: z numbering is from top to bottom, so need to invert
    // so that velocity points from bottom to top

    if ((x < nx) && (y < ny) && (z > 0) && (z < (nz-1))) {

        float * q_p, *q_m, * f;
        q_p = (float *)malloc(vec_dim * sizeof(float));
        q_m = (float *)malloc(vec_dim * sizeof(float));
        f = (float *)malloc(vec_dim * sizeof(float));

        // z-direction
        for (int i = 0; i < vec_dim; i++) {
            float S_upwind = (Un_d[(((z-1) * ny + y) * nx + x) * vec_dim + i] -
                Un_d[((z * ny + y) * nx + x) * vec_dim + i]);
            float S_downwind = (Un_d[((z * ny + y) * nx + x) * vec_dim + i] -
                Un_d[(((z+1) * ny + y) * nx + x) * vec_dim + i]);
            float S = 0.5 * (S_upwind + S_downwind); // S_av

            float r = 1.0e6;

            // make sure don't divide by zero
            if (abs(S_downwind) > 1.0e-7) {
                r = S_upwind / S_downwind;
            }

            S *= phi(r);

            q_p[i] = Un_d[offset + i] + S * 0.5;
            q_m[i] = Un_d[offset + i] - S * 0.5;
        }

        // fluxes
        flux_func(q_p, f, 2, alpha0, gamma, zmin, dz, nz, z, R);

        for (int i = 0; i < vec_dim; i++) {
            qz_plus_half[offset + i] = q_p[i];
            fz_plus_half[offset + i] = f[i];
        }

        flux_func(q_m, f, 2, alpha0, gamma, zmin, dz, nz, z, R);

        for (int i = 0; i < vec_dim; i++) {
            qz_minus_half[offset + i] = q_m[i];
            fz_minus_half[offset + i] = f[i];
        }
        free(q_p);
        free(q_m);
        free(f);
    }
}

__global__ void evolve_fv_fluxes(float * F,
                     float * qx_plus_half, float * qx_minus_half,
                     float * qy_plus_half, float * qy_minus_half,
                     float * fx_plus_half, float * fx_minus_half,
                     float * fy_plus_half, float * fy_minus_half,
                     int nx, int ny, int nz, int vec_dim, float alpha0,
                     float dx, float dy, float dz, float dt, float zmin,
                     float R,
                     int kx_offset, int ky_offset) {
    /**
    Calculates fluxes in finite volume evolution by solving the Riemann
    problem at the cell boundaries.

    Parameters
    ----------
    F : float *
        flux vector at each point in grid and each layer
    qx_plus_half, qx_minus_half : float *
        state vector reconstructed at right and left boundaries
    qy_plus_half, qy_minus_half : float *
        state vector reconstructed at top and bottom boundaries
    fx_plus_half, fx_minus_half : float *
        flux vector at right and left boundaries
    fy_plus_half, fy_minus_half : float *
        flux vector at top and bottom boundaries
    nx, ny, nz : int
        dimensions of grid
    vec_dim : int
        dimension of state vector
    alpha : float
        lapse function
    dx, dy, dt : float
        gridpoint spacing and timestep spacing
    kx_offset, ky_offset : int
        x, y offset for current kernel
    */
    int x = kx_offset + blockIdx.x * blockDim.x + threadIdx.x;
    int y = ky_offset + blockIdx.y * blockDim.y + threadIdx.y;
    int z = threadIdx.z;

    float fx_m, fx_p, fy_m, fy_p;

    // do fluxes
    if ((x > 1) && (x < (nx-2)) && (y > 1) && (y < (ny-2)) && (z < nz)) {
        float alpha;
        if (vec_dim < 6) {
            // shallow water
            alpha = sqrt(exp(-2.0 * 0.25 * (qx_plus_half[((z * ny + y) * nx + x) * vec_dim] + qx_minus_half[((z * ny + y) * nx + x) * vec_dim] + qy_plus_half[((z * ny + y) * nx + x) * vec_dim] + qy_minus_half[((z * ny + y) * nx + x) * vec_dim])));
        } else {
            float h = zmin + dz * (nz - 1 - z);
            float M = 1;
            alpha = alpha0 + M * h / (R*R * alpha0);
        }

        for (int i = 0; i < vec_dim; i++) {
            // x-boundary
            // from i-1
            fx_m = 0.5 * (
                fx_plus_half[((z * ny + y) * nx + x-1) * vec_dim + i] +
                fx_minus_half[((z * ny + y) * nx + x) * vec_dim + i] +
                qx_plus_half[((z * ny + y) * nx + x-1) * vec_dim + i] -
                qx_minus_half[((z * ny + y) * nx + x) * vec_dim + i]);
            // from i+1
            fx_p = 0.5 * (
                fx_plus_half[((z * ny + y) * nx + x) * vec_dim + i] +
                fx_minus_half[((z * ny + y) * nx + x+1) * vec_dim + i] +
                qx_plus_half[((z * ny + y) * nx + x) * vec_dim + i] -
                qx_minus_half[((z * ny + y) * nx + x+1) * vec_dim + i]);

            // y-boundary
            // from j-1
            fy_m = 0.5 * (
                fy_plus_half[((z * ny + y-1) * nx + x) * vec_dim + i] +
                fy_minus_half[((z * ny + y) * nx + x) * vec_dim + i] +
                qy_plus_half[((z * ny + y-1) * nx + x) * vec_dim + i] -
                qy_minus_half[((z * ny + y) * nx + x) * vec_dim + i]);
            // from j+1
            fy_p = 0.5 * (
                fy_plus_half[((z * ny + y) * nx + x) * vec_dim + i] +
                fy_minus_half[((z * ny + y+1) * nx + x) * vec_dim + i] +
                qy_plus_half[((z * ny + y) * nx + x) * vec_dim + i] -
                qy_minus_half[((z * ny + y+1) * nx + x) * vec_dim + i]);

            float old_F = F[((z * ny + y) * nx + x)*vec_dim + i];
            F[((z * ny + y) * nx + x)*vec_dim + i] =
                -alpha * ((fx_p - fx_m)/dx + (fy_p - fy_m)/dy);

            // hack?
            if (nan_check(F[((z * ny + y) * nx + x)*vec_dim + i])) {
                //printf("nan :( (%d, %d, %d) i: %d, fx_p: %f, fx_m: %f, fy_p: %f, fy_m: %f\n", x, y, z, i, fx_p, fx_m, fy_p, fy_m);
                F[((z * ny + y) * nx + x)*vec_dim + i] = old_F;
            }
        }
        //printf("fxm, fxp: %f, %f fym, fyp: %f, %f F(tau): %f\n", fx_m, fx_p, fy_m, fy_p, F[((z * ny + y) * nx + x)*vec_dim +4]);
    }
}

__global__ void evolve_z_fluxes(float * F,
                     float * qz_plus_half, float * qz_minus_half,
                     float * fz_plus_half, float * fz_minus_half,
                     int nx, int ny, int nz, int vec_dim, float alpha0,
                     float dz, float dt, float zmin, float R,
                     int kx_offset, int ky_offset) {
    /**
    Calculates fluxes in finite volume evolution by solving the Riemann
    problem at the cell boundaries in z direction.

    Parameters
    ----------
    F : float *
        flux vector at each point in grid and each layer
    qz_plus_half, qz_minus_half : float *
        state vector reconstructed at right and left boundaries
    fz_plus_half, fz_minus_half : float *
        flux vector at top and bottom boundaries
    nx, ny, nz : int
        dimensions of grid
    vec_dim : int
        dimension of state vector
    alpha : float
        lapse function
    dz, dt : float
        gridpoint spacing and timestep spacing
    kx_offset, ky_offset : int
        x, y offset for current kernel
    */
    int x = kx_offset + blockIdx.x * blockDim.x + threadIdx.x;
    int y = ky_offset + blockIdx.y * blockDim.y + threadIdx.y;
    int z = threadIdx.z;

    // do fluxes
    if ((x > 1) && (x < (nx-2)) && (y > 1) && (y < (ny-2)) &&
        (z > 1) && (z < (nz-2))) {

        float h = zmin + dz * (nz - 1 - z);
        float M = 1;
        float alpha = alpha0 + M * h / (R*R * alpha0);

        // NOTE: z numbering is from top to bottom, so need to invert
        // so that velocity points from bottom to top

        for (int i = 0; i < vec_dim; i++) {
            // z-boundary
            // from below
            float fz_m = 0.5 * (
                fz_plus_half[((z * ny + y) * nx + x) * vec_dim + i] +
                fz_minus_half[(((z+1) * ny + y) * nx + x) * vec_dim + i] +
                qz_plus_half[((z * ny + y) * nx + x) * vec_dim + i] -
                qz_minus_half[(((z+1) * ny + y) * nx + x) * vec_dim + i]);
            // from above
            float fz_p = 0.5 * (
                fz_plus_half[(((z-1) * ny + y) * nx + x) * vec_dim + i] +
                fz_minus_half[((z * ny + y) * nx + x) * vec_dim + i] +
                qz_plus_half[(((z-1) * ny + y) * nx + x) * vec_dim + i] -
                qz_minus_half[((z * ny + y) * nx + x) * vec_dim + i]);

            float old_F = F[((z * ny + y) * nx + x)*vec_dim + i];

            F[((z * ny + y) * nx + x)*vec_dim + i] =
                F[((z * ny + y) * nx + x)*vec_dim + i]
                - alpha * (fz_p - fz_m) / dz;

            // hack?
            if (nan_check(F[((z * ny + y) * nx + x)*vec_dim + i]))
                F[((z * ny + y) * nx + x)*vec_dim + i] = old_F;
        }

        //printf("Fz before: %f, after: %f\n", before, F[((z * ny + y) * nx + x)*vec_dim + 3]);
    }
}

__global__ void grav_sources(float * q, float gamma,
    int nx, int ny, int nz, int vec_dim, float zmin, float R, float alpha0,
    float dz, float dt,
    int kx_offset, int ky_offset) {
    /**
    Calculate gravitational source terms
    **/
    int x = kx_offset + blockIdx.x * blockDim.x + threadIdx.x;
    int y = ky_offset + blockIdx.y * blockDim.y + threadIdx.y;
    int z = threadIdx.z;
    int offset = ((z * ny + y) * nx + x) * vec_dim;

    if ((x > 0) && (x < (nx-1)) &&
        (y > 0) && (y < (ny-1)) &&
        (z > 0) && (z < (nz-1))) {

        float h = zmin + dz * (nz - 1 - z);
        float M = 1;
        float alpha = alpha0 + M * h / (R*R * alpha0);
        float * gamma_up;
        gamma_up = (float *)malloc(9 * sizeof(float));
        for (int i = 0; i < 9; i++) {
            gamma_up[i] = 0.0;
        }
        gamma_up[0] = 1.0;
        gamma_up[4] = 1.0;
        gamma_up[8] = alpha*alpha;

        const float TOL = 1.0e-5;
        float D = q[offset];
        float Sx = q[offset+1];
        float Sy = q[offset+2];
        float Sz = q[offset+3];
        float tau = q[offset+4];

        float Ssq = Sx*Sx*gamma_up[0] + 2.0*Sx*Sy*gamma_up[1] +
            2.0*Sx*Sz*gamma_up[2] + Sy*Sy*gamma_up[4] + 2.0*Sy*Sz*gamma_up[5] +
            Sz*Sz*gamma_up[8];

        float pmin = (1.0 - Ssq) * (1.0 - Ssq) * tau * (gamma - 1.0);
        float pmax = (gamma - 1.0) * (tau + D) / (2.0 - gamma);

        if (pmin < 0.0) {
            pmin = 0.0;//1.0e-9;
        }
        if (pmax < 0.0 || pmax < pmin) {
            pmax = 1.0;
        }

        // check sign change
        if (f_of_p(pmin, D, Sx, Sy, Sz, tau, gamma, gamma_up) *
            f_of_p(pmax, D, Sx, Sy, Sz, tau, gamma, gamma_up) > 0.0) {
            pmin = 0.0;
        }
        if (f_of_p(pmin, D, Sx, Sy, Sz, tau, gamma, gamma_up) *
            f_of_p(pmax, D, Sx, Sy, Sz, tau, gamma, gamma_up) > 0.0) {
            pmax *= 10.0;
        }

        float p = zbrent((fptr)f_of_p, pmin, pmax, TOL, D, Sx, Sy, Sz,
                        tau, gamma, gamma_up);
        if (nan_check(p) || p < 0.0 || p > 1.0e9){
            p = abs((gamma - 1.0) * (tau + D) / (2.0 - gamma)) > 1.0 ? 1.0 :
                abs((gamma - 1.0) * (tau + D) / (2.0 - gamma));
        }

        float sq = sqrt(pow(tau + p + D, 2) - Ssq);
        if (nan_check(sq))
            sq = tau + p + D;

        float hh, W2;

        if (abs(D) < TOL) {
            hh = 1.0;
            W2 = 1.0;
        } else {
            hh = 1.0 + gamma * (sq - p * (tau + p + D)/sq - D) / D;
            W2 = 1.0 + Ssq / (D*D*hh*hh);
        }

        printf("source term/dt: %f, p: %f, Sx, Sy, Sz: (%f, %f, %f), D: %f\n", (-M / (R*R) * (Sz*Sz / W2 + (tau + p + D) / alpha)), p, Sx, Sy, Sz, D);

        q[offset+3] += dt *
            (-M / (R*R) * (Sz*Sz / W2 + (tau + p + D) / alpha));
    }
}

__global__ void evolve_fv_heating(float * Up, float * U_half,
                     float * qx_plus_half, float * qx_minus_half,
                     float * qy_plus_half, float * qy_minus_half,
                     float * fx_plus_half, float * fx_minus_half,
                     float * fy_plus_half, float * fy_minus_half,
                     float * sum_phs, float * rho, float * Q_d,
                     int nx, int ny, int nlayers, float alpha, float gamma,
                     float dx, float dy, float dt,
                     bool burning, float Cv, float E_He,
                     int kx_offset, int ky_offset) {
    /**
    Does the heating part of the evolution.

    Parameters
    ----------
    Up : float *
        state vector at next timestep
    U_half : float *
        state vector at half timestep
    qx_plus_half, qx_minus_half : float *
        state vector reconstructed at right and left boundaries
    qy_plus_half, qy_minus_half : float *
        state vector reconstructed at top and bottom boundaries
    fx_plus_half, fx_minus_half : float *
        flux vector at right and left boundaries
    fy_plus_half, fy_minus_half : float *
        flux vector at top and bottom boundaries
    sum_phs : float *
        sum of Phi in different layers
    rho : float *
        list of densities in different layers
    Q_d : float *
        heating rate in each layer
    nx, ny, nlayers : int
        dimensions of grid
    alpha, gamma : float
        lapse function and adiabatic index
    dx, dy, dt : float
        gridpoint spacing and timestep spacing
    burning : bool
        is burning present in this system?
    Cv, E_He : float
        specific heat in constant volume and energy release per unit mass of He
    kx_offset, ky_offset : int
        x, y offset for current kernel
    */
    int x = kx_offset + blockIdx.x * blockDim.x + threadIdx.x;
    int y = ky_offset + blockIdx.y * blockDim.y + threadIdx.y;
    int z = threadIdx.z;
    int offset = (z * ny + y) * nx + x;

    // calculate Q
    //calc_Q(Up, rho_d, Q_d, nx, ny, nlayers, kx_offset, ky_offset, burning);

    float W = 1.0;

    float X_dot = 0.0;

    // do source terms
    if ((x < nx) && (y < ny) && (z < nlayers)) {
        float * q_swe;
        q_swe = (float *)malloc(4 * sizeof(float));

        for (int i = 0; i < 4; i++) {
            q_swe[i] = U_half[offset * 4 + i];
        }
        float * gamma_up;
        gamma_up = (float *)malloc(9 * sizeof(float));
        for (int i = 0; i < 9; i++) {
            gamma_up[i] = 0.0;
        }
        gamma_up[0] = 1.0;
        gamma_up[4] = 1.0;
        gamma_up[8] = exp(2.0 * q_swe[0]);
        W = W_swe(q_swe, gamma_up);

        float * A, * phis;
        A = (float *)malloc(nlayers * sizeof(float));
        phis = (float *)malloc(nlayers * sizeof(float));
        for (int i = 0; i < nlayers; i++) {
            phis[i] = U_half[((i * ny + y) * nx + x)* 4];
        }

        calc_As(rho, phis, A, nlayers, gamma, phis[0], rho[0]);

        float p = p_from_swe(q_swe, rho[z], gamma, W, A[z]);
        float Y = q_swe[3] / q_swe[0];

        X_dot = calc_Q_swe(rho[z], p, gamma, Y, Cv) / E_He;

        free(phis);
        free(A);
        free(q_swe);
        free(gamma_up);

        U_half[offset*4] /= W;
    }

    __syncthreads();

    if ((x < nx) && (y < ny) && (z < nlayers)) {

        sum_phs[offset] = 0.0;

        float sum_qs = 0.0;
        float deltaQx = 0.0;
        float deltaQy = 0.0;

        if (z < (nlayers - 1)) {
            sum_qs += (Q_d[z + 1] - Q_d[z]);
            deltaQx = Q_d[z] * (U_half[offset*4+1] -
                 U_half[(((z + 1) * ny + y) * nx + x)*4+1]) /
                (W * U_half[offset*4]);
            deltaQy = (Q_d[z]) * (U_half[offset*4+2] -
                 U_half[(((z + 1) * ny + y) * nx + x)*4+2]) /
                (W * U_half[offset*4]);
        }
        if (z > 0) {
            sum_qs += -rho[z-1] / rho[z] * (Q_d[z] - Q_d[z - 1]);
            deltaQx = rho[z-1] / rho[z] * Q_d[z] *
                (U_half[offset*4+1] -
                 U_half[(((z - 1) * ny + y) * nx + x)*4+1]) /
                 (W * U_half[offset*4]);
            deltaQy = rho[z-1] / rho[z] * Q_d[z] *
                (U_half[offset*4+2] -
                 U_half[(((z - 1) * ny + y) * nx + x)*4+2]) /
                 (W * U_half[offset*4]);
        }

        for (int j = 0; j < z; j++) {
            sum_phs[offset] += rho[j] / rho[z] *
                U_half[((j * ny + y) * nx + x)*4];
        }
        for (int j = z+1; j < nlayers; j++) {
            sum_phs[offset] += U_half[((j * ny + y) * nx + x)*4];
        }

        // NOTE: for now going to make Xdot a constant
        //const float X_dot = 0.01;

        // D
        Up[offset*4] += dt * alpha * sum_qs;

        //if (x < 10 && y < 10) printf("(%d, %d, %d) Q: %f, sum_qs: %f, deltaQx: %f, deltaQy: %f\n", x, y, z, Q_d[z], sum_qs, deltaQx, deltaQy);

        // Sx
        Up[offset*4+1] += dt * alpha * (-deltaQx);

        // Sy
        Up[offset*4+2] += dt * alpha * (-deltaQy);

        // DX
        Up[offset*4+3] += dt * alpha * X_dot;
    }
}

__global__ void evolve2(float * Un_d, float * Up, float * U_half,
                     float * sum_phs, int nx, int ny, int nlayers, int ng,
                     float alpha, float dx, float dy, float dt,
                     int kx_offset, int ky_offset) {
    /**
    Adds buoyancy terms.

    Parameters
    ----------
    Un_d : float *
        state vector at each grid point in each layer at current timestep
    Up : float *
        state vector at next timestep
    U_half : float *
        state vector at half timestep
    sum_phs : float *
        sum of Phi in different layers
    nx, ny, nlayers : int
        dimensions of grid
    ng : int
        number of ghost cells
    alpha : float
        lapse function
    dx, dy, dt : float
        gridpoint spacing and timestep spacing
    kx_offset, ky_offset : int
        x, y offset for current kernel
    */
    int x = kx_offset + blockIdx.x * blockDim.x + threadIdx.x;
    int y = ky_offset + blockIdx.y * blockDim.y + threadIdx.y;
    int z = threadIdx.z;
    int offset =  (z * ny + y) * nx + x;

    //printf("kx_offset: %i\n", kx_offset);

    if ((x > 1) && (x < (nx-2)) && (y > 1) && (y < (ny-2)) && (z < nlayers)) {

        float a_upwind = sum_phs[(z * ny + y) * nx + x+1] - sum_phs[offset];
        float a_downwind = sum_phs[offset] - sum_phs[(z * ny + y) * nx + x-1];

        float a = 0.5 * (a_upwind + a_downwind);

        float r = 1.0e6;
        if (abs(a_downwind) > 1.0e-10) {
            r = a_upwind / a_downwind;
        }

        a *= dt * alpha * U_half[offset*4] * 0.5 * phi(r);
        if (abs(a) < 0.9 * dx / dt) {
            Up[offset*4+1] -= a;
        }

        a_upwind = sum_phs[(z * ny + y+1) * nx + x] - sum_phs[offset];
        a_downwind = sum_phs[offset] - sum_phs[(z * ny + y-1) * nx + x];

        a = 0.5 * (a_upwind + a_downwind);

        r = 1.0e6;
        if (abs(a_downwind) > 1.0e-10) {
            r = a_upwind / a_downwind;
        }

        a *= dt * alpha * U_half[offset*4] * 0.5 * phi(r);

        if (abs(a) < 0.9 * dy / dt) {
            Up[offset*4+2] -= a;
        }

        // copy back to grid
        for (int i = 0; i < 4; i++) {
            Un_d[offset*4+i] = Up[offset*4+i];
        }
    }
}

void homogeneuous_fv(dim3 * kernels, dim3 * threads, dim3 * blocks,
       int * cumulative_kernels, float * Un_d, float * F_d,
       float * qx_p_d, float * qx_m_d, float * qy_p_d, float * qy_m_d,
       float * qz_p_d, float * qz_m_d,
       float * fx_p_d, float * fx_m_d, float * fy_p_d, float * fy_m_d,
       float * fz_p_d, float * fz_m_d,
       int nx, int ny, int nz, int vec_dim, int ng, float alpha0, float gamma,
       float dx, float dy, float dz, float dt, int rank, float zmin, float R,
       flux_func_ptr h_flux_func, bool do_z) {
    /**
    Solves the homogeneous part of the equation (ie the bit without source terms).

    Parameters
    ----------
    kernels, threads, blocks : dim3 *
        number of kernels, threads and blocks for each process/kernel
    cumulative_kernels : int *
        Cumulative total of kernels in ranks < rank of current MPI process
    Un_d : float *
        state vector at each grid point in each layer at current timestep
    F_d : float *
        flux vector
    qx_p_d, qx_m_d : float *
        state vector reconstructed at right and left boundaries
    qy_p_d, qy_m_d : float *
        state vector reconstructed at top and bottom boundaries
    fx_p_d, fx_m_d : float *
        flux vector at right and left boundaries
    fy_p_d, fy_m_d : float *
        flux vector at top and bottom boundaries
    nx, ny, nz : int
        dimensions of grid
    alpha, gamma : float
        lapse function and adiabatic index
    dx, dy, dz, dt : float
        gridpoint spacing and timestep spacing
    rank : int
        rank of MPI process
    flux_func : flux_func_ptr
        pointer to function to be used to calculate fluxes
    do_z : bool
        should we evolve in the z direction?
    */

    int kx_offset = 0;
    int ky_offset = (kernels[0].y * blocks[0].y * threads[0].y - 2*ng) * rank;

    int k_offset = 0;
    if (rank > 0) {
        k_offset = cumulative_kernels[rank - 1];
    }

    for (int j = 0; j < kernels[rank].y; j++) {
       kx_offset = 0;
       for (int i = 0; i < kernels[rank].x; i++) {
           evolve_fv<<<blocks[k_offset + j * kernels[rank].x + i], threads[k_offset + j * kernels[rank].x + i]>>>(Un_d, h_flux_func,
                  qx_p_d, qx_m_d, qy_p_d, qy_m_d,
                  fx_p_d, fx_m_d, fy_p_d, fy_m_d,
                  nx, ny, nz, vec_dim, alpha0, gamma,
                  zmin, dz, R,
                  kx_offset, ky_offset);
           if (do_z) {
               evolve_z<<<blocks[k_offset + j * kernels[rank].x + i], threads[k_offset + j * kernels[rank].x + i]>>>(Un_d, h_flux_func,
                      qz_p_d, qz_m_d,
                      fz_p_d, fz_m_d,
                      nx, ny, nz, vec_dim, alpha0, gamma,
                      zmin, dz, R,
                      kx_offset, ky_offset);
           }
           kx_offset += blocks[k_offset + j * kernels[rank].x + i].x *
                threads[k_offset + j * kernels[rank].x + i].x - 2*ng;
       }
       ky_offset += blocks[k_offset + j * kernels[rank].x].y *
            threads[k_offset + j * kernels[rank].x].y - 2*ng;
    }

    ky_offset = (kernels[0].y * blocks[0].y * threads[0].y - 2*ng) * rank;

    for (int j = 0; j < kernels[rank].y; j++) {
       kx_offset = 0;
       for (int i = 0; i < kernels[rank].x; i++) {
           evolve_fv_fluxes<<<blocks[k_offset + j * kernels[rank].x + i], threads[k_offset + j * kernels[rank].x + i]>>>(
                  F_d,
                  qx_p_d, qx_m_d, qy_p_d, qy_m_d,
                  fx_p_d, fx_m_d, fy_p_d, fy_m_d,
                  nx, ny, nz, vec_dim, alpha0,
                  dx, dy, dz, dt, zmin, R, kx_offset, ky_offset);

            if (do_z) {
                evolve_z_fluxes<<<blocks[k_offset + j * kernels[rank].x + i], threads[k_offset + j * kernels[rank].x + i]>>>(
                       F_d,
                       qz_p_d, qz_m_d,
                       fz_p_d, fz_m_d,
                       nx, ny, nz, vec_dim, alpha0,
                       dz, dt, zmin, R, kx_offset, ky_offset);
            }

            kx_offset += blocks[k_offset + j * kernels[rank].x + i].x *
                threads[k_offset + j * kernels[rank].x + i].x - 2*ng;
       }
       ky_offset += blocks[k_offset + j * kernels[rank].x].y *
            threads[k_offset + j * kernels[rank].x].y - 2*ng;
    }
}

void rk3(dim3 * kernels, dim3 * threads, dim3 * blocks,
       int * cumulative_kernels,
       float * Un_d, float * F_d, float * Up_d,
       float * qx_p_d, float * qx_m_d, float * qy_p_d, float * qy_m_d,
       float * qz_p_d, float * qz_m_d,
       float * fx_p_d, float * fx_m_d, float * fy_p_d, float * fy_m_d,
       float * fz_p_d, float * fz_m_d,
       int level,
       int *nxs, int *nys, int *nzs, int *vec_dims, int ng, float alpha0, float R, float gamma,
       float dx, float dy, float dz, float dt,
       float * Up_h, float * F_h, float * Un_h,
       MPI_Comm comm, MPI_Status status, int rank, int n_processes,
       flux_func_ptr flux_func, bool do_z, bool periodic,
       int m_in, float * U_swe, int * matching_indices, float zmin) {
    /**
    Integrates the homogeneous part of the ODE in time using RK3.

    Parameters
    ----------
    kernels, threads, blocks : dim3 *
        number of kernels, threads and blocks for each process/kernel
    cumulative_kernels : int *
        Cumulative total of kernels in ranks < rank of current MPI process
    Un_d : float *
        state vector at each grid point in each layer at current timestep on device
    F_d : float *
        flux vector on device
    Up_d : float *
        state vector at next timestep on device
    qx_p_d, qx_m_d : float *
        state vector reconstructed at right and left boundaries
    qy_p_d, qy_m_d : float *
        state vector reconstructed at top and bottom boundaries
    fx_p_d, fx_m_d : float *
        flux vector at right and left boundaries
    fy_p_d, fy_m_d : float *
        flux vector at top and bottom boundaries
    nx, ny, nz : int
        dimensions of grid
    vec_dim : int
        dimension of state vector
    ng : int
        number of ghost cells
    alpha, gamma : float
        lapse function and adiabatic index
    dx, dy, dz, dt : float
        gridpoint spacing and timestep spacing
    Up_h, F_h, Un_h : float *
        state vector at next timestep, flux vector and state vector at current timestep on host
    comm : MPI_Comm
        MPI communicator
    status: MPI_Status
        status of MPI processes
    rank, n_processes : int
        rank of current MPI process and total number of MPI processes
    flux_func : flux_func_ptr
        pointer to function to be used to calculate fluxes
    do_z : bool
        should we evolve in the z direction?
    periodic : bool
        do we use periodic or outflow boundary conditions?
    */
    //cout << "\nu1\n\n\n";
    // u1 = un + dt * F(un)
    homogeneuous_fv(kernels, threads, blocks, cumulative_kernels,
          Un_d, F_d,
          qx_p_d, qx_m_d, qy_p_d, qy_m_d, qz_p_d, qz_m_d,
          fx_p_d, fx_m_d, fy_p_d, fy_m_d, fz_p_d, fz_m_d,
          nxs[level], nys[level], nzs[level], vec_dims[level], ng, alpha0, gamma,
          dx, dy, dz, dt, rank, zmin, R, flux_func, do_z);

    // copy back flux
    hipMemcpy(F_h, F_d, nxs[level]*nys[level]*nzs[level]*vec_dims[level]*sizeof(float),
        hipMemcpyDeviceToHost);
    if (n_processes == 1) {
        bcs_fv(F_h, nxs[level], nys[level], nzs[level], ng, vec_dims[level], periodic, do_z);
        if (do_z) {
            enforce_hse(F_h, U_swe,
                            nxs, nys, nzs, ng,
                            level, m_in, zmin, dz,
                            matching_indices, gamma, R, alpha0);
        }
    } else {
        int y_size = kernels[0].y * blocks[0].y * threads[0].y - 2*ng;
        bcs_mpi(F_h, nxs[level], nys[level], nzs[level], vec_dims[level], ng, comm, status, rank, n_processes,
                y_size, do_z, periodic);
    }

    for (int n = 0; n < nxs[level]*nys[level]*nzs[level]*vec_dims[level]; n++) {
        Up_h[n] = Un_h[n] + dt * F_h[n];
    }
    // enforce boundaries and copy back
    if (n_processes == 1) {
        bcs_fv(Up_h, nxs[level], nys[level], nzs[level], ng, vec_dims[level], periodic, do_z);
        if (do_z) {
            enforce_hse(Up_h, U_swe,
                            nxs, nys, nzs, ng,
                            level, m_in, zmin, dz,
                            matching_indices, gamma, R, alpha0);
        }

    } else {
        int y_size = kernels[0].y * blocks[0].y * threads[0].y - 2*ng;
        bcs_mpi(Up_h, nxs[level], nys[level], nzs[level], vec_dims[level], ng, comm, status, rank,
                n_processes, y_size, do_z, periodic);
    }

    if (do_z) {
        // HACK:
        // going to do some hacky data sanitisation here
        // NOTE: could argue that this is actually a form of artificial
        // dissipation to ensure stability (as it is just smoothing out
        // spikes in the data after all)
        for (int x = 0; x < nxs[level] * nys[level] * nzs[level]; x++) {
            if (abs(Up_h[x*6]) > 1.0e2) {
                Up_h[x*6] = 0.5;
            }
            if (abs(Up_h[x*6+4]) > 1.0e3 || Up_h[x*6+4] < 0.0) {
                Up_h[x*6+4] = Up_h[x*6];
            }
            if (Up_h[x*6+5] > 1.0) Up_h[x*6+5] = 1.0;
            if (Up_h[x*6+5] < 0.0) Up_h[x*6+5] = 0.0;
            for (int i = 1; i < 4; i++) {
                if (abs(Up_h[x*6+i]) > Up_h[x*6]) {
                    Up_h[x*6+i] = 0.0;
                }
            }
        }
    }

    hipMemcpy(Un_d, Up_h, nxs[level]*nys[level]*nzs[level]*vec_dims[level]*sizeof(float),
               hipMemcpyHostToDevice);
    //cout << "\nu2\n\n\n";
    // u2 = 0.25 * (3*un + u1 + dt*F(u1))
    homogeneuous_fv(kernels, threads, blocks, cumulative_kernels,
          Un_d, F_d,
          qx_p_d, qx_m_d, qy_p_d, qy_m_d, qz_p_d, qz_m_d,
          fx_p_d, fx_m_d, fy_p_d, fy_m_d, fz_p_d, fz_m_d,
          nxs[level], nys[level], nzs[level], vec_dims[level], ng, alpha0, gamma,
          dx, dy, dz, dt, rank, zmin, R, flux_func, do_z);

    // copy back flux
    hipMemcpy(F_h, F_d, nxs[level]*nys[level]*nzs[level]*vec_dims[level]*sizeof(float),
               hipMemcpyDeviceToHost);

    if (n_processes == 1) {
        bcs_fv(F_h, nxs[level], nys[level], nzs[level], ng, vec_dims[level], periodic, do_z);
        if (do_z) {
            enforce_hse(F_h, U_swe,
                            nxs, nys, nzs, ng,
                            level, m_in, zmin, dz,
                            matching_indices, gamma, R, alpha0);
        }
    } else {
        int y_size = kernels[0].y * blocks[0].y * threads[0].y - 2*ng;
        bcs_mpi(F_h, nxs[level], nys[level], nzs[level], vec_dims[level], ng, comm, status, rank, n_processes,
                y_size, do_z, periodic);
    }

    for (int n = 0; n < nxs[level]*nys[level]*nzs[level]*vec_dims[level]; n++) {
        Up_h[n] = 0.25 * (3.0 * Un_h[n] + Up_h[n] + dt * F_h[n]);
    }

    // enforce boundaries and copy back
    if (n_processes == 1) {
        bcs_fv(Up_h, nxs[level], nys[level], nzs[level], ng, vec_dims[level], periodic, do_z);
        if (do_z) {
            enforce_hse(Up_h, U_swe,
                            nxs, nys, nzs, ng,
                            level, m_in, zmin, dz,
                            matching_indices, gamma, R, alpha0);
        }
    } else {
        int y_size = kernels[0].y * blocks[0].y * threads[0].y - 2*ng;
        bcs_mpi(Up_h, nxs[level], nys[level], nzs[level], vec_dims[level], ng, comm, status, rank,
                n_processes, y_size, do_z, periodic);
    }

    if (do_z) {
        // HACK:
        // going to do some hacky data sanitisation here
        for (int x = 0; x < nxs[level] * nys[level] * nzs[level]; x++) {
            if (abs(Up_h[x*6]) > 1.0e2) {
                Up_h[x*6] = 0.5;
            }
            if (abs(Up_h[x*6+4]) > 1.0e3 || Up_h[x*6+4] < 0.0) {
                Up_h[x*6+4] = Up_h[x*6];
            }
            if (Up_h[x*6+5] > 1.0) Up_h[x*6+5] = 1.0;
            if (Up_h[x*6+5] < 0.0) Up_h[x*6+5] = 0.0;
            for (int i = 1; i < 4; i++) {
                if (abs(Up_h[x*6+i]) > Up_h[x*6]) {
                    Up_h[x*6+i] = 0.0;
                }
            }
        }
    }

    hipMemcpy(Un_d, Up_h, nxs[level]*nys[level]*nzs[level]*vec_dims[level]*sizeof(float),
               hipMemcpyHostToDevice);
    //cout << "\nun+1\n\n\n";
    // un+1 = (1/3) * (un + 2*u2 + 2*dt*F(u2))
    homogeneuous_fv(kernels, threads, blocks, cumulative_kernels,
          Un_d, F_d,
          qx_p_d, qx_m_d, qy_p_d, qy_m_d, qz_p_d, qz_m_d,
          fx_p_d, fx_m_d, fy_p_d, fy_m_d, fz_p_d, fz_m_d,
          nxs[level], nys[level], nzs[level], vec_dims[level], ng, alpha0, gamma,
          dx, dy, dz, dt, rank, zmin, R, flux_func, do_z);

    // copy back flux
    hipMemcpy(F_h, F_d, nxs[level]*nys[level]*nzs[level]*vec_dims[level]*sizeof(float),
               hipMemcpyDeviceToHost);

    if (n_processes == 1) {
        bcs_fv(F_h, nxs[level], nys[level], nzs[level], ng, vec_dims[level], periodic, do_z);
        if (do_z) {
            enforce_hse(F_h, U_swe,
                            nxs, nys, nzs, ng,
                            level, m_in, zmin, dz,
                            matching_indices, gamma, R, alpha0);
        }
    } else {
        int y_size = kernels[0].y * blocks[0].y * threads[0].y - 2*ng;
        bcs_mpi(F_h, nxs[level], nys[level], nzs[level], vec_dims[level], ng, comm, status, rank, n_processes,
                y_size, do_z, periodic);
    }

    for (int n = 0; n < nxs[level]*nys[level]*nzs[level]*vec_dims[level]; n++) {
        Up_h[n] = (1/3.0) * (Un_h[n] + 2.0*Up_h[n] + 2.0*dt * F_h[n]);
    }

    // enforce boundaries
    if (n_processes == 1) {
        bcs_fv(Up_h, nxs[level], nys[level], nzs[level], ng, vec_dims[level], periodic, do_z);
        if (do_z) {
            enforce_hse(Up_h, U_swe,
                            nxs, nys, nzs, ng,
                            level, m_in, zmin, dz,
                            matching_indices, gamma, R, alpha0);
        }
    } else {
        int y_size = kernels[0].y * blocks[0].y * threads[0].y - 2*ng;
        bcs_mpi(Up_h, nxs[level], nys[level], nzs[level], vec_dims[level], ng, comm, status, rank,
                n_processes, y_size, do_z, periodic);
    }

    if (do_z) {
        // HACK: going to do some hacky data sanitisation here
        for (int x = 0; x < nxs[level] * nys[level] * nzs[level]; x++) {
            if (abs(Up_h[x*6]) > 1.0e2) {
                Up_h[x*6] = 0.5;
            }
            if (abs(Up_h[x*6+4]) > 1.0e3 || Up_h[x*6+4] < 0.0) {
                Up_h[x*6+4] = Up_h[x*6];
            }
            if (Up_h[x*6+5] > 1.0) Up_h[x*6+5] = 1.0;
            if (Up_h[x*6+5] < 0.0) Up_h[x*6+5] = 0.0;
            for (int i = 1; i < 4; i++) {
                if (abs(Up_h[x*6+i]) > Up_h[x*6]) {
                    Up_h[x*6+i] = 0.0;
                }
            }
        }
    }

    for (int j = 0; j < nxs[level]*nys[level]*nzs[level]*vec_dims[level]; j++) {
        //if (!do_z) Un_h[j] = Up_h[j];
        Un_h[j] = Up_h[j];
    }
}

template<typename T>
T array_max(T * a, int length) {
    // Returns the maximum value of array a
    T max = a[0];
    for (int i = 1; i < length; i++) {
        if (a[i] > max) max = a[i];
    }
    return max;
}
